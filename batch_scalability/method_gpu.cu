#include "hip/hip_runtime.h"
#include "data.cuh"
#include "util.cuh"
#include <random>

namespace BatchScalability
{

__global__ void kernelGpuNaive1(Data data)
{
    uint64_t threadId = (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    int reqIdx = threadId / data.numDocs;
    int docIdx = threadId % data.numDocs;
    if (reqIdx < data.numReqs && docIdx < data.numDocs)
    {
        double rst = 0;
        for (int embIdx = 0; embIdx < data.embDim; embIdx++)
        {
            float reqVal = data.d_reqData[getMemAddrReq(reqIdx, embIdx, data.numReqs, data.embDim)];
            float docVal = data.d_docData[getMemAddrDoc(docIdx, embIdx, data.numDocs, data.embDim)];
            rst += std::sqrt(reqVal * docVal);
        }
        data.d_rstDataGpu[getMemAddrRst(reqIdx, docIdx, data.numReqs, data.numDocs)] = rst;    
    }
}

void methodGpuNaive1(Data& data)
{
    uint64_t blockSize = kBlockSize;
    uint64_t gridSize = (data.numReqs * data.numDocs + blockSize - 1) / blockSize;
    kernelGpuNaive1<<<gridSize, blockSize>>>(data);
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError());
}

__global__ void kernelGpuNaive2(Data data)
{
    int threadId = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    int reqIdx = threadId % data.numReqs;
    int docIdx = threadId / data.numReqs;
    double rst = 0;
    for (int embIdx = 0; embIdx < data.embDim; embIdx++)
    {
        float reqVal = data.d_reqData[getMemAddrReq(reqIdx, embIdx, data.numReqs, data.embDim)];
        float docVal = data.d_docData[getMemAddrDoc(docIdx, embIdx, data.numDocs, data.embDim)];
        rst += std::sqrt(reqVal * docVal);
    }
    data.d_rstDataGpu[getMemAddrRst(reqIdx, docIdx, data.numReqs, data.numDocs)] = rst;
}

void methodGpuNaive2(Data& data)
{
    uint64_t blockSize = kBlockSize;
    uint64_t gridSize = (data.numReqs * data.numDocs + blockSize - 1) / blockSize;
    kernelGpuNaive2<<<gridSize, blockSize>>>(data);
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError());
}

__global__ void kernelGpuNaive3(Data data)
{
    int reqIdx = blockIdx.y;
    int docIdx = blockIdx.x;
    double rst = 0;
    for (int embIdx = 0; embIdx < data.embDim; embIdx++)
    {
        float reqVal = data.d_reqData[getMemAddrReq(reqIdx, embIdx, data.numReqs, data.embDim)];
        float docVal = data.d_docData[getMemAddrDoc(docIdx, embIdx, data.numDocs, data.embDim)];
        rst += std::sqrt(reqVal * docVal);
    }
    data.d_rstDataGpu[getMemAddrRst(reqIdx, docIdx, data.numReqs, data.numDocs)] = rst;
}

void methodGpuNaive3(Data& data)
{
    dim3 blockSize(1024 / data.numReqs, data.numReqs);
    dim3 gridSize((data.numDocs + blockSize.x - 1) / blockSize.x, 1);
    kernelGpuNaive3<<<gridSize, blockSize>>>(data);
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError());
}

} // namespace BatchScalability