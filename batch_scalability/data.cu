#include <hip/hip_runtime.h>
#include "data.cuh"

#include "util.cuh"

namespace BatchScalability {

Data genData(int numReqs, int numDocs, int embDim)
{
    Data data;
    data.numReqs = numReqs;
    data.numDocs = numDocs;
    data.embDim = embDim;
    CHECK_CUDA(hipMallocManaged(&data.d_docData, numDocs * embDim * sizeof(float)));
    CHECK_CUDA(hipMallocManaged(&data.d_reqData, numReqs * embDim * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&data.h_rstDataCpu, numDocs * numReqs * sizeof(float) ));
    CHECK_CUDA(hipMallocManaged(&data.d_rstDataGpu, numDocs * numReqs * sizeof(float) ));
    return data;
}

void freeData(Data& data)
{
    if (data.d_docData != nullptr) {
        hipFree(data.d_docData);
    }
    if (data.d_reqData != nullptr) {
        hipFree(data.d_reqData);
    }
    if (data.h_rstDataCpu != nullptr) {
        hipHostFree(data.h_rstDataCpu);
    }
    if (data.d_rstDataGpu != nullptr) {
        hipFree(data.d_rstDataGpu);
    }
}

} // namespace BatchScalability