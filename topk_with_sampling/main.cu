#include <random>
#include <vector>
#include <stdexcept>
#include <iostream>

#include "common.cuh"
#include "topk.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "[main.cu] CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

int kNumToRetrieve = 10;
int kNumTrials = 1;

void runExp(int numReqs, int numDocs)
{
    cout << "\n\nrunning exps with numReq: " << numReqs << ", numDocs: " << numDocs << endl;

    TopkParam param;
    param.numReqs = numReqs;
    param.numDocs = numDocs;
    param.numToRetrieve = kNumToRetrieve;

    CHECK_CUDA(hipMallocManaged(&param.dm_score, numDocs * numReqs * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&param.hp_rstCpu, numReqs * kNumToRetrieve * sizeof(Pair)));
    CHECK_CUDA(hipMallocManaged(&param.dm_rstGpu, numReqs * kNumToRetrieve * sizeof(Pair)));

    default_random_engine generator;
    uniform_real_distribution<float> distribution(-1.0, 1.0);
    for (size_t i = 0; i < numDocs * numReqs; i++)
    {
        param.dm_score[i] = distribution(generator);
    }

    TopkSampling topkSampling;
    topkSampling.malloc();

    for (int t = -3; t < kNumTrials; t++)
    {
        retrieveTopkCpu(param);
        topkSampling.retrieveTopk(param);

        for (int reqIdx = 0; reqIdx < numReqs; reqIdx++)
        {
            for (int docIdx = 0; docIdx < kNumToRetrieve; docIdx++)
            {
                size_t memAddr = getMemAddr(reqIdx, docIdx, kNumToRetrieve);
                Pair cpuPair = param.hp_rstCpu[memAddr];
                Pair gpuPair = param.dm_rstGpu[memAddr];
                if (cpuPair.docId != gpuPair.docId || cpuPair.score != gpuPair.score)
                {
                    cout << "Error: CPU and GPU results do not match" << endl;
                    break;
                }
            }
        }
    }

    CHECK_CUDA(hipFree(param.dm_score));
    CHECK_CUDA(hipHostFree(param.hp_rstCpu));
    CHECK_CUDA(hipFree(param.dm_rstGpu));
}

int main()
{
    runExp(1, 100);

    return 0;
}