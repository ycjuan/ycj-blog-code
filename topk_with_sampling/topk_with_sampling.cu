#include <vector>
#include <cmath>
#include <functional>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <cassert>
#include <numeric>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <string>

#include "topk.cuh"
#include "common.cuh"
#include "util.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "[topk_with_bucket_sort.cu] CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

void TopkSampling::init()
{

}

void TopkSampling::reset()
{

}

void TopkSampling::retrieveTopk(TopkParam &param)
{
    CudaTimer timerTotal;
    CudaTimer timerApprox;
    timerTotal.tic();
    timerApprox.tic();

    // Step1 - Sample
    sample(param);

    // Step2 - Sort
    float threshold = 0;
    findThreshold(param, threshold);

    // Step3 - Copy eligible 
    size_t numCopied = 0;
    copyEligible(param, threshold, numCopied);
    param.gpuApproxTimeMs = timerApprox.tocMs();

    // Step4 - retreiveExact
    retrieveExact(param);

    param.gpuTotalTimeMs = timerTotal.tocMs();
}
