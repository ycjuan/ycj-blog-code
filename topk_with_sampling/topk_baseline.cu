#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <algorithm>
#include <omp.h>

#include "common.cuh"
#include "topk.cuh"
#include "util.cuh"


using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "[topk_baseline.cu] CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

void retrieveTopkCpu(TopkParam &param)
{
    CudaTimer timer;
    timer.tic();

    omp_set_num_threads(8);
    #pragma omp parallel for
    for (int j = 0; j < param.numReqs; j++)
    {
        vector<Pair> v_doc;
        for (int i = 0; i < param.numDocs; i++)
        {
            Pair doc;
            doc.reqIdx = j;
            doc.docIdx = i;
            doc.score = param.dm_score[j * param.numDocs + i];
            v_doc.push_back(doc);
        }

        stable_sort(v_doc.begin(), v_doc.end(), scoreComparator);
        for (int i = 0; i < param.numToRetrieve; i++)
        {
            param.hp_rstCpu[j * param.numToRetrieve + i] = v_doc[i];
        }

        if (j % 8 == 0)
        {
            cout << "retrieved topk for req " << j << endl;
        }
    }

    param.cpuTimeMs = timer.tocMs();
}
