#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <algorithm>

#include "common.cuh"
#include "topk.cuh"
#include "util.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "[topk_baseline.cu] CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

void retrieveTopkCpu(TopkParam &param)
{
    CudaTimer timer;
    timer.tic();

    for (int j = 0; j < param.numReqs; j++)
    {
        vector<Pair> v_doc;
        for (int i = 0; i < param.numDocs; i++)
        {
            Pair doc;
            doc.reqId = j;
            doc.docId = i;
            doc.score = param.dm_score[j * param.numDocs + i];
            v_doc.push_back(doc);
        }

        sort(v_doc.begin(), v_doc.end(), scoreComparator);
        for (int i = 0; i < param.numToRetrieve; i++)
        {
            param.hp_rstCpu[j * param.numToRetrieve + i] = v_doc[i];
        }
    }

    param.cpuTimeMs = timer.tocMs();
}
