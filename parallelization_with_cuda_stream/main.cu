#include "hip/hip_runtime.h"
#include "util.cuh"
#include <bits/types/struct_sched_param.h>
#include <cassert>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

struct Config
{
    int numReqs;
    int numDocs;
    int numRepeats;
    int numTrials;

    const int kBlockSize = 1024;

    void print()
    {
        std::cout << "numReqs: " << numReqs << std::endl;
        std::cout << "numDocs: " << numDocs << std::endl;
        std::cout << "numRepeats: " << numRepeats << std::endl;
        std::cout << "numTrials: " << numTrials << std::endl;
    }
};

std::vector<std::vector<long>> getCpuReference(Config config)
{
    std::vector<std::vector<long>> rst2D(config.numReqs, std::vector<long>(config.numDocs));
    #pragma omp parallel for
    for (int reqIdx = 0; reqIdx < config.numReqs; reqIdx++)
    {
        for (int docIdx = 0; docIdx < config.numDocs; docIdx++)
        {
            long rst = docIdx + reqIdx * config.numDocs;
            for (int i = 0; i < config.numRepeats; i++)
            {
                rst = (rst * 1103515245 + 12345) & 0x7fffffff;
                rst = (rst >> 3) ^ (rst << 7);
                rst = (rst * 16807) % 2147483647;
            }
            rst2D[reqIdx][docIdx] = rst;
        }
    }
    return rst2D;
}

__global__ void dummyKernel(long* d_rst, int reqIdx, int numDocs, int numRepeats)
{
    int docIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (docIdx < numDocs)
    {
        long rst = docIdx + reqIdx * numDocs;
        for (int i = 0; i < numRepeats; i++)
        {
            // Some pseudo-random computation to avoid compiler optimization
            rst = (rst * 1103515245 + 12345) & 0x7fffffff;
            rst = (rst >> 3) ^ (rst << 7);
            rst = (rst * 16807) % 2147483647;            
        }
        d_rst[docIdx + reqIdx * numDocs] = rst;
    }
}

void runReqByReq(Config config, const std::vector<std::vector<long>>& cpuReference)
{
    // ----------------
    // Preparation
    long* d_rst;
    CHECK_CUDA(hipMalloc(&d_rst, config.numReqs * config.numDocs * sizeof(long)));

    // ----------------
    // Run experiment
    Timer timer;
    for (int t = -3; t < config.numTrials; t++)
    {
        if (t == 0)
            timer.tic();
        for (int reqIdx = 0; reqIdx < config.numReqs; reqIdx++)
        {
            int gridSize = (int)ceil((double)(config.numDocs + 1) / config.kBlockSize);
            dummyKernel<<<gridSize, config.kBlockSize>>>(d_rst, reqIdx, config.numDocs, config.numRepeats);
        }
    }
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipGetLastError());
    float timeMs = timer.tocMs() / config.numTrials;
    std::cout << "Time taken for req by req: " << timeMs << " ms" << std::endl;

    // ----------------
    // Check correctness
    long *h_rst;
    CHECK_CUDA(hipHostMalloc(&h_rst, config.numReqs * config.numDocs * sizeof(long)));
    CHECK_CUDA(hipMemcpy(h_rst, d_rst, config.numReqs * config.numDocs * sizeof(long), hipMemcpyDeviceToHost));
    for (int reqIdx = 0; reqIdx < config.numReqs; reqIdx++)
    {
        for (int docIdx = 0; docIdx < config.numDocs; docIdx++)
        {
            assert(h_rst[reqIdx * config.numDocs + docIdx] == cpuReference[reqIdx][docIdx]);
        }
    }
    std::cout << "All results are correct ^____^" << std::endl;

    // ----------------
    // Cleanup
    CHECK_CUDA(hipFree(d_rst));
    CHECK_CUDA(hipHostFree(h_rst));
}

void runParallelWithCudaStream(Config config, int numCudaStreams, const std::vector<std::vector<long>>& cpuReference)
{
    // ----------------
    // Preparation
    long* d_rst;
    CHECK_CUDA(hipMalloc(&d_rst, config.numReqs * config.numDocs * sizeof(long)));
    std::vector<hipStream_t> streams;
    for (int i = 0; i < numCudaStreams; i++)
    {
        hipStream_t stream;
        CHECK_CUDA(hipStreamCreate(&stream));
        streams.push_back(stream);
    }

    // ----------------
    // Run experiment
    Timer timer;

    for (int t = -3; t < config.numTrials; t++)
    {
        if (t == 0)
            timer.tic();
        for (int reqIdx = 0; reqIdx < config.numReqs; reqIdx++)
        {
            int gridSize = (int)ceil((double)(config.numDocs + 1) / config.kBlockSize);
            dummyKernel<<<gridSize, config.kBlockSize, 0, streams[reqIdx % numCudaStreams]>>>(
                d_rst, reqIdx, config.numDocs, config.numRepeats);
        }
        for (int i = 0; i < numCudaStreams; i++)
        {
            CHECK_CUDA(hipStreamSynchronize(streams[i]));
        }
    }
    float timeMs = timer.tocMs() / config.numTrials;
    std::cout << "Time taken with " << numCudaStreams << " cuda streams: " << timeMs << " ms" << std::endl;

    // ----------------
    // Check correctness
    long *h_rst;
    CHECK_CUDA(hipHostMalloc(&h_rst, config.numReqs * config.numDocs * sizeof(long)));
    CHECK_CUDA(hipMemcpy(h_rst, d_rst, config.numReqs * config.numDocs * sizeof(long), hipMemcpyDeviceToHost));
    for (int reqIdx = 0; reqIdx < config.numReqs; reqIdx++)
    {
        for (int docIdx = 0; docIdx < config.numDocs; docIdx++)
        {
            assert(h_rst[reqIdx * config.numDocs + docIdx] == cpuReference[reqIdx][docIdx]);
        }
    }
    std::cout << "All results are correct ^____^" << std::endl;

    // ----------------
    // Cleanup
    CHECK_CUDA(hipFree(d_rst));
    CHECK_CUDA(hipHostFree(h_rst));
}

void runOneConfig(Config config)
{
    const auto cpuReference = getCpuReference(config);

    runReqByReq(config, cpuReference);

    for (int numCudaStreams = 1; numCudaStreams <= config.numReqs; numCudaStreams *= 2)
    {
        assert(config.numReqs % numCudaStreams == 0);
        runParallelWithCudaStream(config, numCudaStreams, cpuReference);
    }
}

int main()
{
    printDeviceInfo();
    
    {
        Config config;
        config.numReqs = 32;
        config.numDocs = 1000000;
        config.numRepeats = 1000;
        config.numTrials = 100;
        config.print();

        runOneConfig(config);
    }

    std::cout << "\n--------------------------------" << std::endl;

    {
        Config config;
        config.numReqs = 16384;
        config.numDocs = 1000;
        config.numRepeats = 100;
        config.numTrials = 100;
        config.print();

        runOneConfig(config);
    }

    std::cout << "\n--------------------------------" << std::endl;

    {
        Config config;
        config.numReqs = 16384;
        config.numDocs = 1000;
        config.numRepeats = 1000;
        config.numTrials = 100;
        config.print();

        runOneConfig(config);
    }
}