#include "hip/hip_runtime.h"
#include "util.cuh"
#include <bits/types/struct_sched_param.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

constexpr int kNumReqs = 32;
constexpr int kNumDocs = 32;
constexpr int kNumTrials = 1000;
constexpr int kBlockSize = 1024;

__global__ void dummyKernel(long *d_rst, int reqIdx)
{
    int docIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (docIdx < kNumDocs)
    {
        int rst = docIdx + reqIdx * kNumDocs;
        for (int i = 0; i < kNumTrials; i++)
        {
            rst = (rst * 1103515245 + 12345) & 0x7fffffff;\
        }
        d_rst[docIdx + reqIdx * kNumDocs] = rst;
    }
}

void runReqByReq()
{
    // ----------------
    // Preparation
    long *d_rst;
    CHECK_CUDA(hipMalloc(&d_rst, kNumReqs * kNumDocs * sizeof(long)));

    // ----------------
    // Run experiment
    Timer timer;
    timer.tic();
    for (int reqIdx = 0; reqIdx < kNumReqs; reqIdx++)
    {
        int gridSize = (int)ceil((double)(kNumDocs + 1) / kBlockSize);
        dummyKernel<<<gridSize, kBlockSize>>>(d_rst, reqIdx);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());
    }
    float timeMs = timer.tocMs();
    std::cout << "Time taken for req by req: " << timeMs << " ms" << std::endl;

    // ----------------
    // Cleanup
    CHECK_CUDA(hipFree(d_rst));
}

void runParallelWithCudaStream()
{
    // ----------------
    // Preparation
    long *d_rst;
    CHECK_CUDA(hipMalloc(&d_rst, kNumReqs * kNumDocs * sizeof(long)));
    std::vector<hipStream_t> streams;
    for (int i = 0; i < kNumReqs; i++)
    {
        hipStream_t stream;
        CHECK_CUDA(hipStreamCreate(&stream));
        streams.push_back(stream);
    }

    // ----------------
    // Run experiment
    Timer timer;
    timer.tic();
    for (int reqIdx = 0; reqIdx < kNumReqs; reqIdx++)
    {   
        int gridSize = (int)ceil((double)(kNumDocs + 1) / kBlockSize);
        dummyKernel<<<gridSize, kBlockSize, 0, streams[reqIdx]>>>(d_rst, reqIdx);
    }
    for (int i = 0; i < kNumReqs; i++)
    {
        CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }
    float timeMs = timer.tocMs();
    std::cout << "Time taken for parallel with cuda stream: " << timeMs << " ms" << std::endl;

    // ----------------
    // Cleanup
    CHECK_CUDA(hipFree(d_rst));
    for (int i = 0; i < kNumReqs; i++)
    {
        CHECK_CUDA(hipStreamDestroy(streams[i]));
    }
}

int main()
{
    printDeviceInfo();

    runReqByReq();

    runParallelWithCudaStream();

    return 0;
}