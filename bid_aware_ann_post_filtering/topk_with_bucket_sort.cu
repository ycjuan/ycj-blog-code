#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <functional>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <cassert>
#include <numeric>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <string>

#include "topk.cuh"
#include "common.cuh"
#include "util.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "[topk_with_bucket_sort.cu] CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

void Topk::init(float minScore, float maxScore)
{
    minScore_ = minScore;
    maxScore_ = maxScore;
    CHECK_CUDA(hipMalloc(&d_counter_, kSize_byte_d_counter_));
    CHECK_CUDA(hipHostMalloc(&h_counter_, kSize_byte_d_counter_));
}

void Topk::reset()
{
    CHECK_CUDA(hipFree(d_counter_));
    CHECK_CUDA(hipHostFree(h_counter_));
}

__global__ void updateCounterKernel(ReqDocPair *d_doc, int numReqDocPairs, Topk retriever)
{
    int docId = blockIdx.x * blockDim.x + threadIdx.x;

    if (docId < numReqDocPairs)
    {
        ReqDocPair doc = d_doc[docId];
        retriever.updateCounter(doc);
    }
}

void Topk::findLowestBucket(vector<int> &v_counter, int numToRetrieve, int &lowestBucket, int &numReqDocPairsGreaterThanLowestBucket)
{
    lowestBucket = 0;
    numReqDocPairsGreaterThanLowestBucket = 0;
    // Starting from the highest bucket, accumulate the count until it satisfies numToRetrieve
    for (int bucket = kGranularity_; bucket >= 0; bucket--)
    {
        // Accumulate the count of all slots into the first slot
        int slot0 = 0;
        int counterIdx0 = getCounterIdx(slot0, bucket);
        for (int slot = 1; slot < kNumSlots_; slot++)
        {
            int counterIdx = getCounterIdx(slot, bucket);
            v_counter[counterIdx0] += v_counter[counterIdx];
        }
        numReqDocPairsGreaterThanLowestBucket += v_counter[counterIdx0];
        if (numReqDocPairsGreaterThanLowestBucket >= numToRetrieve)
        {
            lowestBucket = bucket;
            break;
        }
    }
}

vector<ReqDocPair> Topk::retrieveTopk(ReqDocPair *d_doc, ReqDocPair *d_buffer, int numReqDocPairs, int numToRetrieve, float &timeMs)
{
    CudaTimer timer;
    timer.tic();

    int kBlockSize = 256;
    int gridSize = (int)ceil((double)(numReqDocPairs + 1) / kBlockSize);

    // Step1 - Run kernel to update the counter
    CHECK_CUDA(hipMemset(d_counter_, 0, kSize_byte_d_counter_))
    updateCounterKernel<<<gridSize, kBlockSize>>>(d_doc, numReqDocPairs, *this);
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError())

    // Step2 - Copy counter from GPU to CPU
    vector<int> v_counter(kSize_d_counter_, 0);
    CHECK_CUDA(hipMemcpy(v_counter.data(), d_counter_, kSize_byte_d_counter_, hipMemcpyDeviceToHost))

    // Step3 - Find the lowest bucket
    int numReqDocPairsGreaterThanLowestBucket;
    findLowestBucket(v_counter, numToRetrieve, lowestBucket_, numReqDocPairsGreaterThanLowestBucket);

    // Step4 - Filter items that is larger than the lowest bucket
    ReqDocPair *d_endPtr = thrust::copy_if(thrust::device, d_doc, d_doc + numReqDocPairs, d_buffer, *this); // copy_if will call Topk::operator()
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError())
    int numCopied = (d_endPtr - d_buffer);
    assert(numCopied == numReqDocPairsGreaterThanLowestBucket);

    // Step5 - Only sort the docs that are larger than the lowest bucket
    thrust::stable_sort(thrust::device, d_buffer, d_buffer + numCopied, ScorePredicator());
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError())

    // Step6 - copy back to CPU
    vector<ReqDocPair> v_doc(numToRetrieve);
    CHECK_CUDA(hipMemcpy(v_doc.data(), d_buffer, sizeof(ReqDocPair) * numToRetrieve, hipMemcpyDeviceToHost))

    timeMs = timer.tocMs();

    return v_doc;
}
