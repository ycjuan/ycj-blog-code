#include "hip/hip_runtime.h"
#include <stdexcept>
#include <random>

#include "data_struct.cuh"

void AbmDataGpu::init(const std::vector<std::vector<std::vector<long>>> &data3D, bool useManagedMemory)
{
    // -----------------
    // Check empty
    {
        if (data3D.empty())
        {
            throw std::runtime_error("data3D is empty");
        }
    
    }

    // -----------------
    // Infer meta data
    {
        int numRows = data3D.size();
        int numFields = data3D.at(0).size();
        int maxNumValsPerRow = 0;

        for (const auto &data2D : data3D)
        {
            if (data2D.size() != numFields)
            {
                throw std::runtime_error("data2D has different number of fields");
            }

            int numValsPerRow = 0;
            for (const auto &data1D : data2D)
            {
                numValsPerRow += data1D.size();
            }
            maxNumValsPerRow = std::max(maxNumValsPerRow, numValsPerRow);
        }
        
        m_maxNumValsPerRow = maxNumValsPerRow;    
        m_numRows = numRows;
        m_numFields = numFields;
    }

    // -----------------
    // Malloc data
    {   
        // -----------
        // Calculate the size of the data
        m_d_data_size = m_numRows * m_maxNumValsPerRow;
        m_d_data_size_in_bytes = m_d_data_size * sizeof(long);
        m_d_offsets_size = m_numRows * (m_numFields + 1);
        m_d_offsets_size_in_bytes = m_d_offsets_size * sizeof(uint32_t);

        // -----------
        // Malloc data
        hipError_t hipError_t;
        if (useManagedMemory)
        {
            hipError_t = hipMallocManaged(&m_d_data, m_d_data_size_in_bytes);
        }
        else
        {
            hipError_t = hipMalloc(&m_d_data, m_d_data_size_in_bytes);
        }
        if (hipError_t != hipSuccess)
        {
            throw std::runtime_error("hipMalloc failed (data): " + std::string(hipGetErrorString(hipError_t)));
        }
        
        if (useManagedMemory)
        {
            hipError_t = hipMallocManaged(&m_d_offsets, m_d_offsets_size_in_bytes);
        }
        else
        {
            hipError_t = hipMalloc(&m_d_offsets, m_d_offsets_size_in_bytes);
        }
        if (hipError_t != hipSuccess)
        {
            throw std::runtime_error("hipMalloc failed (offsets): " + std::string(hipGetErrorString(hipError_t)));
        }
    }

    // -----------------
    // Init data
    {
        // -----------------
        // Malloc pinned memory
        // TODO: Release resource when there is an exception
        long *hp_data;
        hipError_t hipError_t = hipHostMalloc(&hp_data, m_d_data_size_in_bytes);
        if (hipError_t != hipSuccess)
        {
            throw std::runtime_error("hipHostMalloc failed (data): " + std::string(hipGetErrorString(hipError_t)));
        }
        uint32_t *hp_offsets;
        hipError_t = hipHostMalloc(&hp_offsets, m_d_offsets_size_in_bytes);
        if (hipError_t != hipSuccess)
        {
            throw std::runtime_error("hipHostMalloc failed (offsets): " + std::string(hipGetErrorString(hipError_t)));
        }

        // -----------------
        // Fill the data in pinned memory
        for (int row = 0; row < m_numRows; row++)
        {
            int offset = 0;
            hp_offsets[getMemAddrOffsets_dh(row, 0)] = offset;
            for (int field = 0; field < m_numFields; field++)
            {
                for (auto val : data3D.at(row).at(field))
                {
                    hp_data[getMemAddrData_dh(row, offset)] = val;
                    offset++;
                }
                hp_offsets[getMemAddrOffsets_dh(row, field+1)] = offset;
            }
            hp_offsets[getMemAddrOffsets_dh(row, m_numFields)] = offset;
        }

        // -----------------
        // Copy data to device
        hipError_t = hipMemcpy(m_d_data, hp_data, m_d_data_size_in_bytes, hipMemcpyHostToDevice);
        if (hipError_t != hipSuccess)
        {
            throw std::runtime_error("hipMemcpy failed (data): " + std::string(hipGetErrorString(hipError_t)));
        }
        hipError_t = hipMemcpy(m_d_offsets, hp_offsets, m_d_offsets_size_in_bytes, hipMemcpyHostToDevice);
        if (hipError_t != hipSuccess)
        {
            throw std::runtime_error("hipMemcpy failed (offsets): " + std::string(hipGetErrorString(hipError_t)));
        }

        // -----------------
        // Free pinned memory
        hipError_t = hipHostFree(hp_data);
        if (hipError_t != hipSuccess)
        {
            throw std::runtime_error("hipHostFree failed (data): " + std::string(hipGetErrorString(hipError_t)));
        }
        hipError_t = hipHostFree(hp_offsets);
        if (hipError_t != hipSuccess)
        {
            throw std::runtime_error("hipHostFree failed (offsets): " + std::string(hipGetErrorString(hipError_t)));
        }
    }
}

void AbmDataGpu::free()
{
    if (m_d_data != nullptr)
    {
        hipFree(m_d_data);
        m_d_data = nullptr;
    }
    if (m_d_offsets != nullptr)
    {
        hipFree(m_d_offsets);
        m_d_offsets = nullptr;
    }
    m_d_data_size = 0;
    m_d_data_size_in_bytes = 0;
    m_d_offsets_size = 0;
    m_d_offsets_size_in_bytes = 0;
    m_numRows = 0;
    m_numFields = 0;
    m_maxNumValsPerRow = 0;
}

std::vector<std::vector<std::vector<long>>>
genRandData3D(int numRows, int numFields, std::vector<int> numValsPerFieldMin, std::vector<int> numValsPerFieldMax)
{
    // -----------------
    // Check input
    if (numValsPerFieldMin.size() != numFields)
    {
        throw std::runtime_error("numValsPerFieldMin.size() != numFields");
    }
    if (numValsPerFieldMax.size() != numFields)
    {
        throw std::runtime_error("numValsPerFieldMax.size() != numFields");
    }

    // -----------------
    // Prepare random number generator
    std::default_random_engine generator;
    std::uniform_int_distribution<long> valDist;

    // -----------------
    // Generate random data
    std::vector<std::vector<std::vector<long>>> data3D;
    for (int row = 0; row < numRows; row++)
    {
        std::vector<std::vector<long>> data2D;
        for (int field = 0; field < numFields; field++)
        {
            std::vector<long> data1D;
            std::uniform_int_distribution<int> numValsDist(numValsPerFieldMin.at(field), numValsPerFieldMax.at(field));
            int numVals = numValsDist(generator);
            for (int val = 0; val < numVals; val++)
            {
                data1D.push_back(valDist(generator));
            }
            data2D.push_back(data1D);
        }
        data3D.push_back(data2D);
    }

    return data3D;
}