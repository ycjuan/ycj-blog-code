#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>
#include <math.h>
#include <ostream>
#include <sstream>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include "cabm.cuh"
#include "common.cuh"

// We use uint64_t to store the bit stack, so the max number of elements is 64
constexpr uint32_t g_kMaxBitStackCount = 64;

__device__ void stackPushTrue(uint64_t& bitStack, const uint8_t currBitStackIdx)
{
    uint64_t mask = 1L << currBitStackIdx;
    bitStack = bitStack | mask;
}

__device__ void stackPushFalse(uint64_t& bitStack, const uint8_t currBitStackIdx)
{
    uint64_t mask = ~(1L << currBitStackIdx);
    bitStack = bitStack & mask;
}

__device__ bool stackTop(const uint64_t bitStack, const uint8_t currBitStackIdx)
{
    uint64_t mask = 1L << currBitStackIdx;
    uint64_t tmp = bitStack & mask;
    return tmp > 0L;
}

__device__ bool matchOp(const AbmDataGpu& reqAbmDataGpu,
                        const AbmDataGpu& docAbmDataGpu,
                        const int reqIdx,
                        const int docIdx,
                        const CabmOp& op)
{
    int reqOffsetIter = reqAbmDataGpu.getOffset_d(reqIdx, op.getReqFieldIdx());
    int docOffsetIter = docAbmDataGpu.getOffset_d(docIdx, op.getDocFieldIdx());
    int reqOffsetEnd = reqAbmDataGpu.getOffset_d(reqIdx, op.getReqFieldIdx() + 1);
    int docOffsetEnd = docAbmDataGpu.getOffset_d(docIdx, op.getDocFieldIdx() + 1);

    while (reqOffsetIter < reqOffsetEnd && docOffsetIter < docOffsetEnd)
    {
        long reqVal = reqAbmDataGpu.getVal_d(reqIdx, reqOffsetIter);
        long docVal = docAbmDataGpu.getVal_d(docIdx, docOffsetIter);
        if (reqVal == docVal)
        {
            return true;
        }
        else if (reqVal < docVal)
        {
            reqOffsetIter++;
        }
        else
        {
            docOffsetIter++;
        }
    }

    return false;
}

struct OperandKernelParam
{
    AbmDataGpu reqAbmDataGpu;
    AbmDataGpu docAbmDataGpu;
    CabmOp op;
    uint64_t reqIdx;
    uint64_t numDocs;
    uint64_t* d_bitStacks;
    uint8_t* d_bitStackCounts;
};

__global__ void matchOpKernel(OperandKernelParam param)
{
    uint64_t docIdx = (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (docIdx < param.numDocs)
    {
        bool rst = matchOp(param.reqAbmDataGpu, param.docAbmDataGpu, param.reqIdx, docIdx, param.op);
        if (rst)
        {
            stackPushTrue(param.d_bitStacks[docIdx], param.d_bitStackCounts[docIdx]);
        }
        else
        {
            stackPushFalse(param.d_bitStacks[docIdx], param.d_bitStackCounts[docIdx]);
        }
    }
}

struct OperatorKernelParam
{
    CabmOp op;
    uint64_t numPostfixOps;
    uint64_t* d_bitStacks;
    uint8_t* d_bitStackCounts;
    uint64_t numDocs;
};

__global__ void operatorKernel(OperatorKernelParam param)
{
    uint64_t docIdx = (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (docIdx < param.numDocs)
    {
        uint64_t& bitStack = param.d_bitStacks[docIdx];
        uint8_t& bitStackCount = param.d_bitStackCounts[docIdx];
        bool rst1 = stackTop(bitStack, bitStackCount);
        bool rst2 = stackTop(bitStack, bitStackCount);
        bool rst = (param.op.getOpType() == CabmOpType::OPERATOR_AND) ? (rst1 & rst2) : (rst1 | rst2);
        if (rst)
        {
            stackPushTrue(bitStack, bitStackCount);
        }
        else
        {
            stackPushFalse(bitStack, bitStackCount);
        }
    }
}

__global__ void copyRstKernel(uint8_t* d_rst, uint64_t* d_bitStacks, uint64_t numDocs)
{
    uint64_t docIdx = (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (docIdx < numDocs)
    {
        d_rst[docIdx] = stackTop(d_bitStacks[docIdx], 0);
    }
}

struct CabmGpuParam
{
    AbmDataGpu reqAbmDataGpu;
    AbmDataGpu docAbmDataGpu;
    std::vector<CabmOp> postfixOps;
    uint64_t* d_bitStacks;
    uint8_t* d_bitStackCounts;
    uint64_t numDocs;
    uint64_t numReqs;
    uint8_t* d_rst;
};

void cabmGpu(CabmGpuParam param)
{
    const int kBlockSize = 1024;
    const int kGridSize = (param.numDocs + kBlockSize - 1) / kBlockSize;

    for (uint32_t reqIdx = 0; reqIdx < param.numReqs; reqIdx++)
    {
        uint8_t currBitStackIdx = 0;
        CHECK_CUDA(hipMemset(param.d_bitStacks, 0, param.numDocs * sizeof(uint64_t)));
        CHECK_CUDA(hipMemset(param.d_bitStackCounts, 0, param.numDocs * sizeof(uint8_t)));
        for (const auto& op : param.postfixOps)
        {
            if (op.isOperand())
            {
                if (currBitStackIdx >= g_kMaxBitStackCount)
                {
                    std::ostringstream oss;
                    oss << "currBitStackIdx is greater than g_kMaxBitStackCount: " << currBitStackIdx
                        << " >= " << g_kMaxBitStackCount;
                    throw std::runtime_error(oss.str());
                }

                OperandKernelParam operandKernelParam;
                operandKernelParam.reqAbmDataGpu = param.reqAbmDataGpu;
                operandKernelParam.docAbmDataGpu = param.docAbmDataGpu;
                operandKernelParam.op = op;
                operandKernelParam.reqIdx = reqIdx;
                operandKernelParam.numDocs = param.numDocs;
                operandKernelParam.d_bitStacks = param.d_bitStacks;
                operandKernelParam.d_bitStackCounts = param.d_bitStackCounts;
                if (op.getOpType() == CabmOpType::OPERAND_MATCH)
                {
                    matchOpKernel<<<kGridSize, kBlockSize>>>(operandKernelParam);
                }
                else
                {
                    assert(false);
                }
                currBitStackIdx++;
            }
            else if (op.isOperator())
            {
                currBitStackIdx -= 2;

                OperatorKernelParam operatorKernelParam;
                operatorKernelParam.op = op;
                operatorKernelParam.d_bitStacks = param.d_bitStacks;
                operatorKernelParam.d_bitStackCounts = param.d_bitStackCounts;
                operatorKernelParam.numDocs = param.numDocs;
                operatorKernel<<<kGridSize, kBlockSize>>>(operatorKernelParam);
            }
            CHECK_CUDA(hipDeviceSynchronize())
            CHECK_CUDA(hipGetLastError())
        }

        if (currBitStackIdx != 0)
        {
            std::ostringstream oss;
            oss << "currBitStackIdx is not 0: " << currBitStackIdx;
            throw std::runtime_error(oss.str());
        }

        copyRstKernel<<<kGridSize, kBlockSize>>>(param.d_rst, param.d_bitStacks, param.numDocs);
        CHECK_CUDA(hipDeviceSynchronize())
        CHECK_CUDA(hipGetLastError())
    }
}

bool evaluatePostfixGpuWrapped(std::vector<CabmOp> postfix1D,
                               const std::vector<std::vector<long>>& reqData2D,
                               const std::vector<std::vector<long>>& docData2D)
{
    AbmDataGpu reqAbmDataGpu;
    AbmDataGpu docAbmDataGpu;
    reqAbmDataGpu.init({reqData2D}, true);
    docAbmDataGpu.init({docData2D}, true);

    uint8_t* d_rst;
    CHECK_CUDA(hipMallocManaged(&d_rst, 1 * sizeof(uint8_t)));
    uint64_t* d_bitStacks;
    CHECK_CUDA(hipMallocManaged(&d_bitStacks, 1 * sizeof(uint64_t)));
    uint8_t* d_bitStackCounts;
    CHECK_CUDA(hipMallocManaged(&d_bitStackCounts, 1 * sizeof(uint8_t)));

    CabmGpuParam param;
    param.d_rst = d_rst;
    param.d_bitStacks = d_bitStacks;
    param.d_bitStackCounts = d_bitStackCounts;
    param.numDocs = 1;
    param.numReqs = 1;
    param.postfixOps = postfix1D;
    param.reqAbmDataGpu = reqAbmDataGpu;
    param.docAbmDataGpu = docAbmDataGpu;

    cabmGpu(param);

    uint8_t rst = d_rst[0];

    CHECK_CUDA(hipFree(d_rst));
    CHECK_CUDA(hipFree(d_bitStacks));
    CHECK_CUDA(hipFree(d_bitStackCounts));

    return rst;
}