#include <cassert>
#include <cstdint>
#include <iostream>
#include <vector>

#include "data_struct.cuh"
#include "cabm.cuh"
#include "macro.cuh"
#include "util.cuh"

void test3a()
{
    const int kNumReqs = 1;
    const uint64_t kNumDocs = 1000000;
    const int kNumFields = 6;
    const int kNumTrials = 100;
    const std::vector<int> kNumValsPerFieldMin = { 2, 2, 2, 2, 2, 2 };
    const std::vector<int> kNumValsPerFieldMax = { 10, 10, 10, 10, 10, 10 };

    const auto reqData3D = genRandData3D(kNumReqs, kNumFields, kNumValsPerFieldMin, kNumValsPerFieldMax);
    const auto docData3D = genRandData3D(kNumDocs, kNumFields, kNumValsPerFieldMin, kNumValsPerFieldMax);

    std::vector<CabmOp> infix = {
        CabmOp(CabmOpType::LEFT_PARENTHESIS),
        CabmOp(0, 0, CabmOpType::OPERAND_MATCH),
        CabmOp(CabmOpType::OPERATOR_OR),
        CabmOp(1, 1, CabmOpType::OPERAND_MATCH),
        CabmOp(CabmOpType::RIGHT_PARENTHESIS),
        CabmOp(CabmOpType::OPERATOR_AND),
        CabmOp(CabmOpType::LEFT_PARENTHESIS),
        CabmOp(CabmOpType::LEFT_PARENTHESIS),
        CabmOp(3, 3, CabmOpType::OPERAND_MATCH, true),
        CabmOp(CabmOpType::OPERATOR_OR),
        CabmOp(4, 4, CabmOpType::OPERAND_MATCH),
        CabmOp(CabmOpType::RIGHT_PARENTHESIS),
        CabmOp(CabmOpType::OPERATOR_AND),
        CabmOp(5, 5, CabmOpType::OPERAND_MATCH),
        CabmOp(CabmOpType::RIGHT_PARENTHESIS),
    };

    std::vector<CabmOp> postfix = infix2postfix(infix);

    const auto rst2D = cabmCpu(infix, reqData3D, docData3D);

    {
        std::vector<AbmDataGpuOneField> reqAbmDataGpuList;
        std::vector<AbmDataGpuOneField> docAbmDataGpuList;
        for (int fieldIdx = 0; fieldIdx < kNumFields; fieldIdx++)
        {
            reqAbmDataGpuList.push_back(AbmDataGpuOneField());
            docAbmDataGpuList.push_back(AbmDataGpuOneField());
            reqAbmDataGpuList.at(fieldIdx).init({reqData3D}, fieldIdx, true);
            docAbmDataGpuList.at(fieldIdx).init({docData3D}, fieldIdx, true);
        }

        uint8_t* d_rst;
        CHECK_CUDA(hipMalloc(&d_rst, kNumReqs * kNumDocs * sizeof(uint8_t)));
        uint64_t* d_bitStacks;
        CHECK_CUDA(hipMalloc(&d_bitStacks, kNumDocs * sizeof(uint64_t)));

        CabmGpuParam param;
        param.d_rst = d_rst;
        param.d_bitStacks = d_bitStacks;
        param.numDocs = kNumDocs;
        param.numReqs = kNumReqs;
        param.postfixOps = postfix;
        param.reqAbmDataGpuList = reqAbmDataGpuList;
        param.docAbmDataGpuList = docAbmDataGpuList;

        Timer timer;
        float timeMsOperandKernel = 0;
        float timeMsOperatorKernel = 0;
        float timeMsCopyRstKernel = 0;
        float timeMsTotal = 0;
        float timeMsTotalOuter = 0;
        for (int trial = -3; trial < kNumTrials; trial++)
        {
            if (trial == 0)
            {
                timer.tic();
            }
            cabmGpu(param);
            if (trial >= 0)
            {
                timeMsOperandKernel += param.timeMsOperandKernel;
                timeMsOperatorKernel += param.timeMsOperatorKernel;
                timeMsCopyRstKernel += param.timeMsCopyRstKernel;
                timeMsTotal += param.timeMsTotal;
            }
        }
        timeMsTotalOuter = timer.tocMs() / kNumTrials;
        timeMsOperandKernel /= kNumTrials;
        timeMsOperatorKernel /= kNumTrials;
        timeMsCopyRstKernel /= kNumTrials;
        timeMsTotal /= kNumTrials;
        std::cout << "Time total outer: " << timeMsTotalOuter << " ms" << std::endl;
        std::cout << "Time total inner: " << timeMsTotal << " ms" << std::endl;
        std::cout << "Time operand kernel: " << timeMsOperandKernel << " ms" << std::endl;
        std::cout << "Time operator kernel: " << timeMsOperatorKernel << " ms" << std::endl;
        std::cout << "Time copy rst kernel: " << timeMsCopyRstKernel << " ms" << std::endl;

        std::vector<uint8_t> rstGpu(kNumReqs * kNumDocs);
        CHECK_CUDA(hipMemcpy(rstGpu.data(), d_rst, kNumReqs * kNumDocs * sizeof(uint8_t), hipMemcpyDeviceToHost));

        CHECK_CUDA(hipFree(d_rst));
        CHECK_CUDA(hipFree(d_bitStacks));

        std::vector<std::vector<uint8_t>> rstGpu2D(kNumReqs, std::vector<uint8_t>(kNumDocs));
        for (int reqIdx = 0; reqIdx < kNumReqs; reqIdx++)
        {
            for (int docIdx = 0; docIdx < kNumDocs; docIdx++)
            {
                rstGpu2D.at(reqIdx).at(docIdx) = rstGpu.at(reqIdx * kNumDocs + docIdx);
            }
        }
        assert(rst2D == rstGpu2D);
    }
}

int main()
{
    test3a();
    return 0;
}