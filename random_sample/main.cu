#include "hip/hip_runtime.h"
#include <random>
#include <vector>
#include <stdexcept>
#include <iostream>
#include <cassert>
#include <algorithm>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>

#include "util.cuh"

const int kNumDocs = 1000000;
const int kNumTrials = 10;
const int kBlockSize = 512;
const float kSampleRate = 0.1;

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

struct Doc
{
    int docIdx;
    bool isSelected;
};

namespace pseudoRandomCopyIf
{
    __global__ void sampleKernelPseudoRand(Doc *d_docSrc, Doc *d_docDst, int numDocs, int seed, int invSampleRate)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < numDocs)
        {
            Doc &doc = d_docSrc[i];
            int randNum = doc.docIdx + seed;
            doc.isSelected = (randNum % invSampleRate == 0);
        }
    }

    struct Predicator
    {
        __host__ __device__ bool operator()(const Doc x)
        {
            return x.isSelected;
        }
    };

    void sampleFuncPseudoRandCopyIf(Doc *d_docSrc, Doc *d_docDst, int numDocs, float sampleRate)
    {
        int gridSize = (int)ceil((double)(kNumDocs + 1) / kBlockSize);
        int invSampleRate = 1.0 / sampleRate;
        double timeMs = 0;
        for (int t = -3; t < kNumTrials; t++)
        {
            CudaTimer timer;
            timer.tic();
            sampleKernelPseudoRand<<<gridSize, kBlockSize>>>(d_docSrc, d_docDst, numDocs, t, invSampleRate);
            hipDeviceSynchronize();
            CHECK_CUDA(hipGetLastError());
            Doc *d_endPtr = thrust::copy_if(thrust::device, d_docSrc, d_docSrc + numDocs, d_docDst, Predicator());
            int numCopied = d_endPtr - d_docDst;
            cout << "[sampleFuncPseudoRandCopyIf] " << "t: " << t << "numCopied: " << numCopied << endl;
            if (t >= 0)
                timeMs += timer.tocMs();
        }
        timeMs /= kNumTrials;
        cout << "[sampleFuncPseudoRandCopyIf] timeMs: " << timeMs << " ms" << endl;
    }
}
__managed__ int currIdx = 0;

__global__ void sampleKernelPseudoRandWithAtomicAdd(Doc *d_docSrc, Doc *d_docDst, int numDocs, int seed, int invSampleRate)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numDocs)
    {
        Doc &doc = d_docSrc[i];
        int randNum = doc.docIdx + seed;
        doc.isSelected = (randNum % invSampleRate == 0);
        if (doc.isSelected)
        {
            int idx = atomicAdd(&currIdx, 1);
            d_docDst[idx] = doc;
        }
    }
}

void sampleFuncPseudoRandAtomicAdd(Doc *d_docSrc, Doc *d_docDst, int numDocs, float sampleRate)
{
    int gridSize = (int)ceil((double)(kNumDocs + 1) / kBlockSize);
    int invSampleRate = 1.0 / sampleRate;
    double timeMs = 0;
    for (int t = -3; t < kNumTrials; t++)
    {
        CudaTimer timer;
        timer.tic();
        currIdx = 0;
        sampleKernelPseudoRandWithAtomicAdd<<<gridSize, kBlockSize>>>(d_docSrc, d_docDst, numDocs, t, invSampleRate);
        hipDeviceSynchronize();
        CHECK_CUDA(hipGetLastError());
        if (t >= 0)
            timeMs += timer.tocMs();
    }
    timeMs /= kNumTrials;
    cout << "[sampleFuncPseudoRandCopyIf] timeMs: " << timeMs << " ms" << endl;
}
//chunkRandom

int main()
{
    cout << "kNumDocs: " << kNumDocs << ", kSampleRate: " << kSampleRate << endl;

    Doc *d_docSrc = nullptr;
    Doc *d_docDst = nullptr;
    CHECK_CUDA(hipMallocManaged(&d_docSrc, kNumDocs * sizeof(Doc)));
    CHECK_CUDA(hipMallocManaged(&d_docDst, kNumDocs * sizeof(Doc)));

    for (int i = 0; i < kNumDocs; i++)
        d_docSrc[i].docIdx = i;

    pseudoRandomCopyIf::sampleFuncPseudoRandCopyIf(d_docSrc, d_docDst, kNumDocs, kSampleRate);

    hipFree(d_docSrc);
    hipFree(d_docDst);

    return 0;
}