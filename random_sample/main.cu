#include "hip/hip_runtime.h"
#include <random>
#include <vector>
#include <stdexcept>
#include <iostream>
#include <cassert>
#include <algorithm>
#include <unordered_set>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>

#include "util.cuh"

using namespace std;

const int kNumTrials = 10;
const int kBlockSize = 512;
const vector<int> kNumDocsPerPartition = {100, 1000, 10000, 100000, 1000000};
const int kNumPartitions = kNumDocsPerPartition.size();
const int kMaxNumDocs = kNumDocsPerPartition[kNumPartitions - 1];
const int kNumDocsTotal = accumulate(kNumDocsPerPartition.begin(), kNumDocsPerPartition.end(), 0);
const int kSampleSize = 250;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

struct Doc
{
    int docIdx;
    int partitionIdx; // for example, a partition can be things like "country", "language", etc.
};

void checkSample(Doc *d_docDst, int numSampled)
{
    cout << "============" << endl;
    cout << "numSampled: " << numSampled << endl;
    vector<vector<Doc>> sample(kNumPartitions);
    for (int i = 0; i < numSampled; i++)
    {
        Doc doc = d_docDst[i];
        sample[doc.partitionIdx].push_back(doc);
    }
    for (int partitionIdx = 0; partitionIdx < kNumPartitions; partitionIdx++)
        cout << "partitionIdx: " << partitionIdx << ", numSampled: " << sample[partitionIdx].size() << endl;
    cout << "============" << endl;
}

namespace classicRandomSample
{
    __global__ void kernel(Doc *d_docBuffer, int *d_sampleIdxBuffer, Doc *d_docDst, int kSampleSize)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < kSampleSize)
        {
            d_docDst[i] = d_docBuffer[d_sampleIdxBuffer[i]];
        }
    }

    struct Predicator
    {
        int partitionIdx;
        Predicator(int partitionIdx) : partitionIdx(partitionIdx) {}

        __host__ __device__ bool operator()(const Doc x)
        {
            return x.partitionIdx == partitionIdx;
        }
    };

    int sample(Doc *d_docSrc, Doc *d_docDst, Doc *d_docBuffer, int *d_sampleIdxBuffer)
    {
        int sampleSizeAgg = 0;
        for (int partitionIdx = 0; partitionIdx < kNumPartitions; partitionIdx++)
        {
            // extract documents of this partition
            int kNumDocs = kNumDocsPerPartition[partitionIdx];
            Doc *d_docBufferEndPtr = thrust::copy_if(thrust::device, d_docSrc, d_docSrc + kNumDocsTotal, d_docBuffer, Predicator(partitionIdx));
            if (d_docBufferEndPtr - d_docBuffer != kNumDocs)
                throw runtime_error("Error: d_docBufferEndPtr - d_docBuffer != kNumDocs");

            // if num docs in this partition is less than kSampleSize, just copy all
            if (kSampleSize >= kNumDocs)
            {
                CHECK_CUDA(hipMemcpy(d_docDst + sampleSizeAgg, d_docBuffer, kNumDocs * sizeof(Doc), hipMemcpyDeviceToDevice))
                sampleSizeAgg += kNumDocs;
                continue;
            }

            // generate random indexes
            default_random_engine generator;
            uniform_int_distribution<int> distribution(0, kNumDocs);
            unordered_set<int> sampledIdxSet;
            while (sampledIdxSet.size() < kSampleSize)
            {
                int idx = distribution(generator);
                sampledIdxSet.insert(idx);
            }

            // do sampling
            int gridSize = (int)ceil((double)(kSampleSize + 1) / kBlockSize);
            kernel<<<gridSize, kBlockSize>>>(d_docBuffer, d_sampleIdxBuffer, d_docDst + sampleSizeAgg, kSampleSize);
            hipDeviceSynchronize();
            CHECK_CUDA(hipGetLastError());
            sampleSizeAgg += kSampleSize;
        }
        return sampleSizeAgg;
    }

    void runExp(Doc *d_docSrc, Doc *d_docDst)
    {
        Doc *d_docBuffer = nullptr;
        int *d_sampleIdxBuffer = nullptr;
        CHECK_CUDA(hipMalloc(&d_docBuffer, kMaxNumDocs * sizeof(Doc)));
        CHECK_CUDA(hipMalloc(&d_sampleIdxBuffer, kSampleSize * sizeof(int)));

        double timeMs = 0;
        for (int t = -3; t < kNumTrials; t++)
        {
            CudaTimer timer;
            timer.tic();
            int sampleSizeAgg = sample(d_docSrc, d_docDst, d_docBuffer, d_sampleIdxBuffer);
            if (t >= 0)
                timeMs += timer.tocMs();
            if (t == 0)
                checkSample(d_docDst, sampleSizeAgg);
        }
        timeMs /= kNumTrials;
        cout << "[classicRandomSample] timeMs: " << timeMs << " ms" << endl;

        hipFree(d_docBuffer);
        hipFree(d_sampleIdxBuffer);
    }
}

namespace adhocRandomSampleGreedy
{
    const int kNumChunks = 1024;
    __managed__ int docDstCurrIdx;

    struct KernelParam
    {
        Doc *d_docSrc;
        Doc *d_docDst;
        int *d_partitionCounter;
        int offset;
        int numDocsPerChunk;
        int randomStartingPoint;
    };

    __global__ void kernel(KernelParam param)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= param.numDocsPerChunk)
            return;
        if (param.offset + i >= param.randomStartingPoint)
            return;
        
        int docIdx = param.offset + i;
        int partitionIdx = param.d_docSrc[docIdx].partitionIdx;

        // below may have read-write race condition. however, it's fine as the consequence is there might be a little bit more than what we want to sample
        if (param.d_partitionCounter[partitionIdx] >= kSampleSize)
            return;
        atomicAdd(param.d_partitionCounter + partitionIdx, 1);
        atomicAdd(&docDstCurrIdx, 1);
        param.d_docDst[docDstCurrIdx] = param.d_docSrc[docIdx];
    }

    int sample(Doc *d_docSrc, Doc *d_docDst, int *d_partitionCounter)
    {
        CHECK_CUDA(hipMemset(d_partitionCounter, 0, kNumPartitions * sizeof(int)))
        default_random_engine generator;
        uniform_int_distribution<int> distribution(0, kNumDocsTotal);
        int randomStartingPoint = distribution(generator);
        int numDocsPerChunk = (int)ceil((double)kNumDocsTotal / kNumChunks);

        int offset = randomStartingPoint;
        docDstCurrIdx = 0;
        for (int chunkIdx = 0; chunkIdx < kNumChunks; chunkIdx++)
        {
            int gridSize = (int)ceil((double)(numDocsPerChunk + 1) / kBlockSize);
            KernelParam param;
            param.d_docSrc = d_docSrc;
            param.d_docDst = d_docDst;
            param.d_partitionCounter = d_partitionCounter;
            param.offset = offset;
            param.numDocsPerChunk = numDocsPerChunk;
            param.randomStartingPoint = randomStartingPoint;
            kernel<<<gridSize, kBlockSize>>>(param);

            bool allPartitionSampled = true;
            for (int partitionIdx = 0; partitionIdx < kNumPartitions; partitionIdx++)
            {
                if (d_partitionCounter[partitionIdx] < kNumDocsPerPartition[partitionIdx])
                {
                    allPartitionSampled = false;
                    break;
                }
            }
            if (allPartitionSampled)
                break;
            
            offset = (offset + numDocsPerChunk) % kNumDocsTotal;
        }

        int numSampled = 0;
        for (int partitionIdx = 0; partitionIdx < kNumPartitions; partitionIdx++)
            numSampled += d_partitionCounter[partitionIdx];
        return numSampled;
    }

    void runExp(Doc *d_docSrc, Doc *d_docDst)
    {
        int *d_partitionCounter = nullptr;
        CHECK_CUDA(hipMalloc(&d_partitionCounter, kSampleSize * sizeof(int)));

        double timeMs = 0;
        for (int t = -3; t < kNumTrials; t++)
        {
            CudaTimer timer;
            timer.tic();
            int sampleSizeAgg = sample(d_docSrc, d_docDst, d_partitionCounter);
            if (t >= 0)
                timeMs += timer.tocMs();
            if (t == 0)
                checkSample(d_docDst, sampleSizeAgg);
        }
        timeMs /= kNumTrials;
        cout << "[classicRandomSample] timeMs: " << timeMs << " ms" << endl;

        hipFree(d_partitionCounter);
    }
}


int main()
{
    cout << "kNumDocsPerPartition: ";
    for (auto kNumDocs : kNumDocsPerPartition)
    {
        cout << kNumDocs << " ";
    }
    cout << endl;
    cout << "kNumDocsTotal: " << kNumDocsTotal << endl;
    cout << "kSampleSize: " << kSampleSize << endl;
    cout << "kNumTrials: " << kNumTrials << endl;
    cout << "kMaxNumDocs: " << kMaxNumDocs << endl;
    cout << "kNumPartitions: " << kNumPartitions << endl;

    Doc *d_docSrc = nullptr;
    Doc *d_docDst = nullptr;
    CHECK_CUDA(hipMallocManaged(&d_docSrc, kNumDocsTotal * sizeof(Doc)));
    CHECK_CUDA(hipMallocManaged(&d_docDst, kNumPartitions * kSampleSize * sizeof(Doc)));

    int docIdx = 0;
    for (int partitionIdx = 0; partitionIdx < kNumPartitions; partitionIdx++)
    {
        int kNumDocs = kNumDocsPerPartition[partitionIdx];
        for (int i = 0; i < kNumDocs; i++)
        {
            d_docSrc[docIdx].docIdx = docIdx;
            d_docSrc[docIdx].partitionIdx = partitionIdx;
            docIdx++;
        }
    }
    if (docIdx != kNumDocsTotal)
        throw runtime_error("Error: docIdx != kNumDocsTotal");

    classicRandomSample::runExp(d_docSrc, d_docDst);

    hipFree(d_docSrc);
    hipFree(d_docDst);

    return 0;
}
