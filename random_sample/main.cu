#include "hip/hip_runtime.h"
#include <random>
#include <vector>
#include <stdexcept>
#include <iostream>
#include <cassert>
#include <algorithm>

#include "util.cuh"

const int kNumDocs = 1000000;
const int kNumTrials = 10;
const int kBlockSize = 512;
const float kSampleRate = 0.1;

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

struct Doc
{
    int docIdx;
    bool isSelected;
};

__global__ void sampleKernelSimpleRandCopyIf(Doc *d_docSrc, Doc *d_docDst, int numDocs, int seed, int invSampleRate)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numDocs)
    {
        Doc &doc = d_docSrc[i];
        int randNum = doc.docIdx + seed;
        doc.isSelected = (randNum % invSampleRate == 0);
    }
}

void sampleFuncSimpleRandCopyIf(Doc *d_docSrc, Doc *d_docDst, int numDocs, float sampleRate)
{
    int gridSize = (int)ceil((double)(kNumDocs + 1) / kBlockSize);
    int invSampleRate = 1.0 / sampleRate;
    double timeMs = 0;
    for (int t = -3; t < kNumTrials; t++)
    {
        CudaTimer timer;
        timer.tic();
        sampleKernelSimpleRandCopyIf<<<gridSize, kBlockSize>>>(d_docSrc, d_docDst, numDocs, invSampleRate, t);
        hipDeviceSynchronize();
        CHECK_CUDA(hipGetLastError());
        if (t >= 0)
            timeMs += timer.tocMs();
    }
    timeMs /= kNumTrials;
    cout << "[sampleFuncSimpleRandCopyIf] timeMs: " << timeMs << " ms" << endl;
}

//chunkRandom

int main()
{
    cout << "kNumDocs: " << kNumDocs << ", kSampleRate: " << kSampleRate << endl;

    Doc *d_docSrc = nullptr;
    Doc *d_docDst = nullptr;
    CHECK_CUDA(hipMallocManaged(&d_docSrc, kNumDocs * sizeof(Doc)));
    CHECK_CUDA(hipMallocManaged(&d_docDst, kNumDocs * sizeof(Doc)));

    for (int i = 0; i < kNumDocs; i++)
        d_docSrc[i].docIdx = i;

    sampleFuncSimpleRandCopyIf(d_docSrc, d_docDst, kNumDocs, kSampleRate);

    hipFree(d_docSrc);
    hipFree(d_docDst);

    return 0;
}