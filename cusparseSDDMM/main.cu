#include "hip/hip_runtime.h"
#include <string>
#include <stdexcept>
#include <iostream>
#include <random>
#include <sstream>
#include <hipblas.h>
#include <type_traits>

#include "util.cuh"
#include "common.cuh"

using namespace std;

int kNumDocs = 1 << 20;
int kNumReqs = 1 << 0;
int kEmbDim = 1 << 10;
int kNumTrials = 100;
float docDensity = 0.1;
MemLayout kMemLayoutDoc = COL_MAJOR;
MemLayout kMemLayoutReq = ROW_MAJOR;
MemLayout kMemLayoutRstCpu = COL_MAJOR;
MemLayout kMemLayoutRstGpuCuda = COL_MAJOR;
MemLayout kMemLayoutRstGpuCublas = COL_MAJOR;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}


template <typename T>
Data<T> genData()
{
    Data<T> data;
    data.numDocs = kNumDocs;
    data.numReqs = kNumReqs;
    data.embDim = kEmbDim;
    data.docMemLayout = kMemLayoutDoc;
    data.reqMemLayout = kMemLayoutReq;
    data.rstLayoutCpu = kMemLayoutRstCpu;
    data.rstLayoutGpuKernel = kMemLayoutRstGpuCuda;
    data.rstLayoutGpuCublas = kMemLayoutRstGpuCublas;
    data.print();
    
    CHECK_CUDA(hipMallocManaged(&data.d_doc, (size_t)data.numDocs * data.embDim * sizeof(T)));
    CHECK_CUDA(hipMallocManaged(&data.d_req, (size_t)data.numReqs * data.embDim * sizeof(T)));
    CHECK_CUDA(hipMallocManaged(&data.d_rst_kernel, (size_t)data.numDocs * data.numReqs * sizeof(float)));
    CHECK_CUDA(hipMallocManaged(&data.d_rst_cublas, (size_t)data.numDocs * data.numReqs * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&data.h_rst_cpu, (size_t)data.numDocs * data.numReqs * sizeof(float)));

    default_random_engine generator;
    uniform_real_distribution<float> distribution(0.0, 1.0);
    for (int i = 0; i < data.numDocs * data.embDim; i++)
        data.d_doc[i] = (T)distribution(generator);
    for (int i = 0; i < data.numReqs * data.embDim; i++)
        data.d_req[i] = (T)distribution(generator);

    return data;
}

template <typename T>
void checkData(Data<T> data)
{
    for (int i = 0; i < data.numDocs; i++)
    {
        for (int j = 0; j < data.numReqs; j++)
        {
            float cpuVal = data.h_rst_cpu[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutCpu)];
            float gpuKernelVal = data.d_rst_kernel[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutGpuKernel)];
            float gpuCublasVal = data.d_rst_cublas[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutGpuCublas)];

            if (abs(cpuVal - gpuKernelVal) / abs(gpuKernelVal) > 1e-3)
            {
                cout << "Kernel error at (" << i << ", " << j << "): " << cpuVal << " != " << gpuKernelVal << endl;
                return;
            }
            
            if (abs(cpuVal - gpuCublasVal) / abs(gpuKernelVal) > 1e-3)
            {
                cout << "Cublas error at (" << i << ", " << j << "): " << cpuVal << " != " << gpuCublasVal << endl;
                return;
            }
        }
    }
}

int main()
{
    Data<T> data = genData<T>();

    methodCpu(data);
    methodCuda(data);

    checkData(data);

    data.free();

    return 0;
}