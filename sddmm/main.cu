#include "hip/hip_runtime.h"
#include <string>
#include <stdexcept>
#include <iostream>
#include <random>
#include <sstream>
#include <hipblas.h>
#include <type_traits>
#include <algorithm>

#include "util.cuh"
#include "common.cuh"
#include "methods.cuh"

using namespace std;

int kNumDocs = 1 << 10;
int kNumReqs = 1 << 4;
int kEmbDim = 1 << 10;
double kDocDensity = 0.1;
int kNumTrials = 10;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

Data genData(Setting setting)
{
    Data data;
    data.numDocs = kNumDocs;
    data.numReqs = kNumReqs;
    data.embDim = kEmbDim;
    data.docMemLayout = setting.docMemLayout;
    data.reqMemLayout = setting.reqMemLayout;
    
    CHECK_CUDA(hipMallocManaged(&data.d_doc, (size_t)data.numDocs * data.embDim * sizeof(T)));
    CHECK_CUDA(hipMallocManaged(&data.d_req, (size_t)data.numReqs * data.embDim * sizeof(T)));
    CHECK_CUDA(hipMallocManaged(&data.d_PairsToScore, (size_t)data.numDocs * data.numReqs * sizeof(Pair)));
    CHECK_CUDA(hipMallocManaged(&data.d_rstCuda, (size_t)data.numDocs * data.numReqs * sizeof(Pair)));
    CHECK_CUDA(hipMallocManaged(&data.d_rstCusparse, (size_t)data.numDocs * data.numReqs * sizeof(Pair)));
    CHECK_CUDA(hipHostMalloc(&data.h_rstCpu, (size_t)data.numDocs * data.numReqs * sizeof(Pair)));

    default_random_engine generator;
    uniform_real_distribution<float> distribution(0.0, 1.0);
    for (int i = 0; i < data.numDocs * data.embDim; i++)
        data.d_doc[i] = (T)distribution(generator);
    for (int i = 0; i < data.numReqs * data.embDim; i++)
        data.d_req[i] = (T)distribution(generator);
    
    data.numPairsToScore = 0;
    for (int reqIdx = 0; reqIdx < data.numReqs; reqIdx++)
    {
        int numEligibleDocsPerReq = int(kDocDensity * data.numDocs);
        vector<int> v_docIdx1D(data.numDocs);
        for (int i = 0; i < data.numDocs; i++)
            v_docIdx1D[i] = i;
        shuffle(v_docIdx1D.begin(), v_docIdx1D.end(), generator);
        sort(v_docIdx1D.begin(), v_docIdx1D.begin() + numEligibleDocsPerReq);
        if (reqIdx < 4)
        {
            cout << "first 10 eligible doc indices for req " << reqIdx << ": ";
            for (int i = 0; i < 10; i++)
                cout << v_docIdx1D[i] << " ";
            cout << endl;
        }
        for (int docIdx = 0; docIdx < numEligibleDocsPerReq; docIdx++)
        {
            Pair pair;
            pair.reqIdx = reqIdx;
            pair.docIdx = v_docIdx1D[docIdx];
            data.d_PairsToScore[data.numPairsToScore++] = pair;
        }
    }
    
    data.print();
    
    return data;
}

void checkData(Data data)
{
    sort(data.h_rstCpu, data.h_rstCpu + data.numPairsToScore, pairComparatorDocFirst);
    sort(data.d_rstCuda, data.d_rstCuda + data.numPairsToScore, pairComparatorDocFirst);
    sort(data.d_rstCusparse, data.d_rstCusparse + data.numPairsToScore, pairComparatorDocFirst);
    for (size_t pairIdx = 0; pairIdx < data.numPairsToScore; pairIdx++)
    {
        Pair pairCpu = data.h_rstCpu[pairIdx];
        Pair pairCuda = data.d_rstCuda[pairIdx];
        Pair pairCusparse = data.d_rstCusparse[pairIdx];

        if (pairCpu.reqIdx != pairCuda.reqIdx ||
            pairCpu.docIdx != pairCuda.docIdx ||
            abs( (pairCpu.score - pairCuda.score) / pairCpu.score) > 1e-3)
        {
            cout << "Mismatch at pairIdx " << pairIdx << endl;
            cout << "CPU: " << pairCpu.reqIdx << " " << pairCpu.docIdx << " " << pairCpu.score << endl;
            cout << "CUDA: " << pairCuda.reqIdx << " " << pairCuda.docIdx << " " << pairCuda.score << endl;
            throw runtime_error("Mismatch detected!");
        }

        if (pairCpu.reqIdx != pairCusparse.reqIdx ||
            pairCpu.docIdx != pairCusparse.docIdx ||
            abs( (pairCpu.score - pairCuda.score) / pairCpu.score) > 1e-3)
        {
            cout << "Mismatch at pairIdx " << pairIdx << endl;
            cout << "CPU: " << pairCpu.reqIdx << " " << pairCpu.docIdx << " " << pairCpu.score << endl;
            cout << "CUSPARSE: " << pairCusparse.reqIdx << " " << pairCusparse.docIdx << " " << pairCusparse.score << endl;
            throw runtime_error("Mismatch detected!");
        }
    }
    cout << "All results match!" << endl;
}

void runExp(Setting setting)
{
    cout << endl << endl;
    setting.print();
    Data data = genData(setting);

    methodCpu(data, setting);
    methodCuda(data, setting);
    methodCusparse(data, setting);

    checkData(data);

    data.free();
}

int main()
{
    vector<MemLayout> docMemLayouts = {ROW_MAJOR, COL_MAJOR};
    vector<MemLayout> reqMemLayouts = {ROW_MAJOR, COL_MAJOR};
    vector<bool> swapDocReqs = {false, true};
    vector<bool> reqFirsts = {false, true};

    Setting setting;
    setting.numTrials = kNumTrials;
    for (auto docMemLayout : docMemLayouts)
    {
        for (auto reqMemLayout : reqMemLayouts)
        {
            for (int i = 0; i < 2; i++)
            {
                setting.docMemLayout = docMemLayout;
                setting.reqMemLayout = reqMemLayout;
                setting.swapDocReq = swapDocReqs[i];
                setting.reqFirst = reqFirsts[i];
                runExp(setting);
            }
        }
    }

    return 0;
}