#include <iostream>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

/* Several code snippets are taken from NVIDIA's cuSPARSE samples:
 *   https://github.com/NVIDIA/CUDALibrarySamples/tree/467734659975dd2d795609bd7c01930cc560338f/cuSPARSE/sddmm_csr
 */ 

#include "common.cuh"
#include "util.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

void methodCusparse(Data data, Setting setting) 
{
    // Host problem definition
    int   A_num_rows   = data.numDocs;
    int   A_num_cols   = data.embDim;
    int   B_num_rows   = A_num_cols;
    int   B_num_cols   = data.numReqs;
    int   C_nnz        = data.numPairsToScore;
    int   lda          = A_num_cols;
    int   ldb          = B_num_cols;
    int   A_size       = lda * A_num_rows;
    int   B_size       = ldb * B_num_rows;
    float alpha        = 1.0f;
    float beta         = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dC_offsets, *dC_columns;
    T *dB, *dA;
    float *dC_values;
    dA = data.d_doc;
    dB = data.d_req;
    CHECK_CUDA( hipMalloc((void**) &dC_offsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_columns, C_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  C_nnz * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dC_offsets, hC_offsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_columns, hC_columns, C_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_values, hC_values, C_nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, A_num_rows, A_num_cols, lda, dA,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // Create sparse matrix C in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, C_nnz,
                                      dC_offsets, dC_columns, dC_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSDDMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    CHECK_CUSPARSE( hipsparseSDDMM_preprocess(
                                  handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // execute SpMM
    CHECK_CUSPARSE( hipsparseSDDMM(handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hC_values, dC_values, C_nnz * sizeof(float),
                           hipMemcpyDeviceToHost) )
    int correct = 1;
    for (int i = 0; i < C_nnz; i++) {
        if (hC_values[i] != hC_result[i]) {
            correct = 0; // direct floating point comparison is not reliable
            break;
        }
    }
    if (correct)
        printf("sddmm_csr_example test PASSED\n");
    else
        printf("sddmm_csr_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC_offsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
    return EXIT_SUCCESS;
}
