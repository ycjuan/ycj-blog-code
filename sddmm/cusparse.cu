/*
 * Copyright 1993-2022 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

/* The code is modified from:
 *   https://github.com/NVIDIA/CUDALibrarySamples/tree/467734659975dd2d795609bd7c01930cc560338f/cuSPARSE/sddmm_csr
 */ 

#include <iostream>
#include <stdexcept>
#include <algorithm>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "common.cuh"
#include "util.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

#define CHECK_CUSPARSE(func)                                                     \
    {                                                                            \
        hipsparseStatus_t status = (func);                                        \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                   \
        {                                                                        \
            string error = "CUSPARSE API failed at line " + to_string(__LINE__); \
            throw runtime_error(error);                                          \
        }                                                                        \
    }

void methodCusparse(Data data, Setting setting) 
{
    // Host problem definition
    int   A_num_rows   = data.numDocs;
    int   A_num_cols   = data.embDim;
    int   B_num_rows   = A_num_cols;
    int   B_num_cols   = data.numReqs;
    int   C_nnz        = data.numPairsToScore;
    int   lda          = A_num_cols;
    int   ldb          = B_num_cols;
    int   A_size       = lda * A_num_rows;
    int   B_size       = ldb * B_num_rows;
    float alpha        = 1.0f;
    float beta         = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dC_offsets, *dC_columns;
    T *dB, *dA;
    float *dC_values;
    dA = data.d_doc;
    dB = data.d_req;
    CHECK_CUDA( hipMallocManaged((void**) &dC_offsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMallocManaged((void**) &dC_columns, C_nnz * sizeof(int))   )
    CHECK_CUDA( hipMallocManaged((void**) &dC_values,  C_nnz * sizeof(float)) )

    coo2Csr(data, dC_offsets, dC_columns, dC_values);


    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, A_num_rows, A_num_cols, lda, dA,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // Create sparse matrix C in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, C_nnz,
                                      dC_offsets, dC_columns, dC_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSDDMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    CHECK_CUSPARSE( hipsparseSDDMM_preprocess(
                                  handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // execute SpMM
    CHECK_CUSPARSE( hipsparseSDDMM(handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------

    size_t pairIdx = 0;
    for (int docIdx = 0; docIdx < data.numDocs; docIdx++)
    {
        int start = dC_offsets[docIdx];
        int end = dC_offsets[docIdx + 1];
        for (int i = start; i < end; i++)
        {
            Pair pair;
            pair.reqIdx = dC_columns[i];
            pair.docIdx = docIdx;
            pair.score = dC_values[i];
            data.d_rstCusparse[pairIdx++] = pair;
        }
    }

    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dC_offsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
}
