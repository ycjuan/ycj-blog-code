/*
 * Copyright 1993-2022 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

/* The code is modified from:
 *   https://github.com/NVIDIA/CUDALibrarySamples/tree/467734659975dd2d795609bd7c01930cc560338f/cuSPARSE/sddmm_csr
 */ 

#include <iostream>
#include <stdexcept>
#include <algorithm>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "common.cuh"
#include "util.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

#define CHECK_CUSPARSE(func)                                                     \
    {                                                                            \
        hipsparseStatus_t status = (func);                                        \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                   \
        {                                                                        \
            string error = "CUSPARSE API failed at line " + to_string(__LINE__); \
            throw runtime_error(error);                                          \
        }                                                                        \
    }

void methodCusparse(Data data, Setting setting) 
{
    // Host problem definition
    int   A_num_rows   = data.numDocs;
    int   A_num_cols   = data.embDim;
    int   B_num_rows   = A_num_cols;
    int   B_num_cols   = data.numReqs;
    int   C_nnz        = data.numPairsToScore;
    int   lda          = (data.docMemLayout == ROW_MAJOR)? A_num_cols : A_num_rows;
    int   ldb          = (data.reqMemLayout == COL_MAJOR)? B_num_cols : B_num_rows;
    int   A_size       = (data.docMemLayout == ROW_MAJOR)? lda * A_num_rows : lda * A_num_cols;
    int   B_size       = (data.reqMemLayout == COL_MAJOR)? ldb * B_num_rows : ldb * B_num_cols;
    float alpha        = 1.0f;
    float beta         = 0.0f;
    cout << "lda = " << lda << ", ldb = " << ldb << endl;
    //--------------------------------------------------------------------------
    int   *hC_offsets, *hC_columns;
    T *hB, *hA;
    float *hC_values;
    // data.d_doc is allocated by hipMallocManaged. however, it seems it doesn't work for cusparse for some reason..
    // that's why I'm treating it as "host memory", and then copy to dA which is allocated by hipMalloc.
    hA = data.d_doc;
    hB = data.d_req;
    CHECK_CUDA( hipHostMalloc((void**) &hC_offsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipHostMalloc((void**) &hC_columns, C_nnz * sizeof(int))   )
    CHECK_CUDA( hipHostMalloc((void**) &hC_values,  C_nnz * sizeof(float)) )
    
    #pragma omp parallel for
    for (size_t pairIdx = 0; pairIdx < data.numPairsToScore; pairIdx++)
        data.d_PairsToScore[pairIdx].score = 0;

    coo2Csr(data, hC_offsets, hC_columns, hC_values);

    // Device memory management
    int   *dC_offsets, *dC_columns;
    T *dB, *dA;
    float *dC_values;
    CHECK_CUDA( hipMalloc((void**) &dA, A_size * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dB, B_size * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC_offsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_offsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_columns, C_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  C_nnz * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA, hA, A_size * sizeof(float),
                           hipMemcpyDeviceToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(float),
                           hipMemcpyDeviceToDevice) )
    CHECK_CUDA( hipMemcpy(dC_offsets, hC_offsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_columns, hC_columns, C_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_values, hC_values, C_nnz * sizeof(float),
                           hipMemcpyHostToDevice) )

    hipDeviceSynchronize();

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A
    hipsparseOrder_t orderA = (data.docMemLayout == ROW_MAJOR)? HIPSPARSE_ORDER_ROW : HIPSPARSE_ORDER_COL;
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, A_num_rows, A_num_cols, lda, dA,
                                        HIP_R_32F, orderA) )
    // Create dense matrix B
    hipsparseOrder_t orderB = (data.reqMemLayout == COL_MAJOR)? HIPSPARSE_ORDER_ROW : HIPSPARSE_ORDER_COL;
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, orderB) )
    // Create sparse matrix C in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, C_nnz,
                                      dC_offsets, dC_columns, dC_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSDDMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    /*
    The doc says:
        The function hipsparseSDDMM_preprocess() can be called before hipsparseSDDMM to speedup the actual computation. 
        It is useful when hipsparseSDDMM is called multiple times with the same sparsity pattern (matC). 
        The values of the dense matrices (matA, matB) can change arbitrarily.
    Since we can't assume "the same sparsity pattern (matC)" for our application, we don't use this function here.
    */
    /*
    CHECK_CUSPARSE( hipsparseSDDMM_preprocess(
                                  handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    */
    // execute SpMM
    Timer timer;
    timer.tic();
    for (int t = -3; t < setting.numTrials; t++)
    {
        /*
        also tried to use this, but doesn't have any effect on the performance.
        CHECK_CUSPARSE(hipsparseSDDMM_preprocess(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, HIP_R_32F,
            HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer))
        */
        CHECK_CUSPARSE(hipsparseSDDMM(handle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                     HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer))
        CHECK_CUDA( hipDeviceSynchronize() )
    }
    cout << "CUSPARSE time: " << timer.tocMs() / setting.numTrials << " ms" << endl;
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    
    CHECK_CUDA( hipMemcpy(hC_values, dC_values, C_nnz * sizeof(float),
                           hipMemcpyDeviceToHost) )
    size_t pairIdx = 0;
    for (int docIdx = 0; docIdx < data.numDocs; docIdx++)
    {
        int start = hC_offsets[docIdx];
        int end = hC_offsets[docIdx + 1];
        for (int i = start; i < end; i++)
        {
            Pair pair;
            pair.reqIdx = hC_columns[i];
            pair.docIdx = docIdx;
            pair.score = hC_values[i];
            data.d_rstCusparse[pairIdx++] = pair;
        }
    }

    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC_offsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
    CHECK_CUDA( hipHostFree(hC_offsets) )
    CHECK_CUDA( hipHostFree(hC_columns) )
    CHECK_CUDA( hipHostFree(hC_values) )
}
