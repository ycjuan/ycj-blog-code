#include "hip/hip_runtime.h"
#include <iostream>

#include "common.cuh"
#include "util.cuh"

using namespace std;

void methodCpu(Data data, Setting setting)
{
    #pragma omp parallel for
    for (size_t pairIdx = 0; pairIdx < data.numPairsToScore; pairIdx++)
        data.d_PairsToScore[pairIdx].score = 0;


    Timer timer;
    timer.tic();
    
    #pragma omp parallel for
    for (size_t pairIdx = 0; pairIdx < data.numPairsToScore; pairIdx++)
    {
        Pair &pair = data.d_PairsToScore[pairIdx];
        pair.score = 0;
        for (int k = 0; k < data.embDim; k++)
        {
            T reqVal = data.d_req[getMemAddr(pair.reqIdx, k, data.numReqs, data.embDim, data.reqMemLayout)];
            T docVal = data.d_doc[getMemAddr(pair.docIdx, k, data.numDocs, data.embDim, data.docMemLayout)];
            pair.score += (float)reqVal * (float)docVal;
        }
    }
    cout << "CPU time: " << timer.tocMs() << " ms" << endl;

    #pragma omp parallel for
    for (size_t pairIdx = 0; pairIdx < data.numPairsToScore; pairIdx++)
        data.h_rstCpu[pairIdx] = data.d_PairsToScore[pairIdx];
}

__global__ void cudaKernel(Data data)
{
    int wid = (size_t)blockIdx.x * blockDim.x + threadIdx.x;

    if (wid < data.numPairsToScore)
    {
        Pair pair = data.d_PairsToScore[wid];
        pair.score = 0;
        for (int k = 0; k < data.embDim; k++)
        {
            T reqVal = data.d_req[getMemAddr(pair.reqIdx, k, data.numReqs, data.embDim, data.reqMemLayout)];
            T docVal = data.d_doc[getMemAddr(pair.docIdx, k, data.numDocs, data.embDim, data.docMemLayout)];
            pair.score += float(reqVal * docVal);
        }
        data.d_PairsToScore[wid] = pair;
    }
}

void methodCuda(Data data, Setting setting)
{
    int blockSize = 512;
    int gridSize = (data.numPairsToScore + blockSize - 1)/ blockSize;

    #pragma omp parallel for
    for (size_t pairIdx = 0; pairIdx < data.numPairsToScore; pairIdx++)
        data.d_PairsToScore[pairIdx].score = 0;
    
    if (setting.reqFirst)
        sort(data.d_PairsToScore, data.d_PairsToScore + data.numPairsToScore, pairComparatorReqFirst);
    else
        sort(data.d_PairsToScore, data.d_PairsToScore + data.numPairsToScore, pairComparatorDocFirst);

    CudaTimer timer;
    for (int t = -3; t < setting.numTrials; t++)
    {
        if (t == 0)
            timer.tic();
        cudaKernel<<<gridSize, blockSize>>>(data);
        hipDeviceSynchronize();
        hipError_t status = hipGetLastError();
        if (status != hipSuccess)
        {
            ostringstream oss;
            oss << "Kernel launch failed with error: " << hipGetErrorString(status) << "\n";
            throw runtime_error(oss.str());
        }
    }
    cout << "Kernel time: " << timer.tocMs() / setting.numTrials << " ms" << endl;

    #pragma omp parallel for
    for (size_t pairIdx = 0; pairIdx < data.numPairsToScore; pairIdx++)
        data.d_rstCuda[pairIdx] = data.d_PairsToScore[pairIdx];
}
