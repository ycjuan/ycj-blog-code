#include "util.cuh"

CudaTimer::CudaTimer()
{
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
}

void CudaTimer::tic()
{
    hipEventRecord(start_);
}

float CudaTimer::tocMs()
{
    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    float elapsedMs;
    hipEventElapsedTime(&elapsedMs, start_, stop_);
    return elapsedMs;
}

CudaTimer::~CudaTimer()
{
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
}
