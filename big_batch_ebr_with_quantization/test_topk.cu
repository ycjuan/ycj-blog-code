#include <random>
#include <vector>
#include <stdexcept>
#include <iostream>

#include "common.cuh"
#include "topk.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "[main.cu] CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

int kNumToRetrieve = 100;
int kNumTrials = 100;

void runExp(int numReqs, int numDocs, bool useRandomSampling, bool useCpu)
{
    cout << "\n\nrunning exps with numReq: " << numReqs << ", numDocs: " << numDocs << endl;

    TopkParam param;
    param.numReqs = numReqs;
    param.numDocs = numDocs;
    param.numToRetrieve = kNumToRetrieve;
    param.useRandomSampling = useRandomSampling;

    size_t allocateInBytes;
    size_t totalAllocateInBytes = 0;
    
    allocateInBytes = (size_t)numDocs * numReqs * sizeof(float);
    CHECK_CUDA(hipMalloc(&param.d_score, allocateInBytes));
    CHECK_CUDA(hipHostMalloc(&param.h_score, allocateInBytes));
    totalAllocateInBytes += allocateInBytes;
    cout << "allocated " << allocateInBytes / 1024.0 / 1024.0 / 1024.0 << " GiB for d_score" << endl;

    allocateInBytes = (size_t)numReqs * kNumToRetrieve * sizeof(Pair);
    CHECK_CUDA(hipHostMalloc(&param.h_rstCpu, allocateInBytes));
    totalAllocateInBytes += allocateInBytes;
    cout << "allocated " << allocateInBytes / 1024.0 / 1024.0 / 1024.0 << " GiB for h_rstCpu" << endl;

    allocateInBytes = (size_t)numReqs * kNumToRetrieve * sizeof(Pair);
    CHECK_CUDA(hipMalloc(&param.d_rstGpu, allocateInBytes));
    totalAllocateInBytes += allocateInBytes;
    cout << "allocated " << allocateInBytes / 1024.0 / 1024.0 / 1024.0 << " GiB for d_rstGpu" << endl;

    cout << "total allocated " << totalAllocateInBytes / 1024.0 / 1024.0 / 1024.0 << " GiB" << endl;

    cout << "initializing scores" << endl;
    default_random_engine generator;
    uniform_real_distribution<float> distribution(-1.0, 1.0);
    for (size_t i = 0; i < numDocs * numReqs; i++)
    {
        param.h_score[i] = distribution(generator);
    }
    CHECK_CUDA(hipMemcpy(param.d_score, param.h_score, numDocs * numReqs * sizeof(float), hipMemcpyHostToDevice));
    cout << "scores initialized" << endl;

    TopkSampling topkSampling;
    topkSampling.malloc();

    double gpuSampleTimeMs = 0;
    double gpuFindThresholdTimeMs = 0;
    double gpuCopyEligibleTimeMs = 0;
    double gpuRetreiveExactTimeMs = 0;
    double gpuTotalTimeMs = 0;
    double gpuApproxTimeMs = 0;

    if (useCpu)
    {
        cout << "retrieving topk with cpu" << endl;
        retrieveTopkCpu(param);
        cout << "topk retrieved with cpu" << endl;
    }
    for (int t = -3; t < kNumTrials; t++)
    {
        topkSampling.retrieveTopk(param);

        if (useCpu && t == -1)
        {
            cout << "compare results" << endl;
            vector<Pair> v_rstGpu(param.numReqs * kNumToRetrieve);
            CHECK_CUDA(hipMemcpy(v_rstGpu.data(), param.d_rstGpu, param.numReqs * kNumToRetrieve * sizeof(Pair), hipMemcpyDeviceToHost));
            for (int reqIdx = 0; reqIdx < numReqs; reqIdx++)
            {
                for (int docIdx = 0; docIdx < kNumToRetrieve; docIdx++)
                {
                    size_t memAddr = getMemAddr(reqIdx, docIdx, kNumToRetrieve);
                    Pair cpuPair = param.h_rstCpu[memAddr];
                    Pair gpuPair = v_rstGpu[memAddr];
                    if (cpuPair.docIdx != gpuPair.docIdx || cpuPair.score != gpuPair.score)
                    {
                        cout << "mismatch at reqIdx: " << reqIdx << ", docIdx: " << docIdx << endl;
                        cout << "cpuPair: " << cpuPair.reqIdx << ", " << cpuPair.docIdx << ", " << cpuPair.score << endl;
                        cout << "gpuPair: " << gpuPair.reqIdx << ", " << gpuPair.docIdx << ", " << gpuPair.score << endl;
                    }
                }
            }
            cout << "results compared!!!" << endl;
        }

        if (t >= 0)
        {
            gpuSampleTimeMs += param.gpuSampleTimeMs;
            gpuFindThresholdTimeMs += param.gpuFindThresholdTimeMs;
            gpuCopyEligibleTimeMs += param.gpuCopyEligibleTimeMs;
            gpuRetreiveExactTimeMs += param.gpuRetreiveExactTimeMs;
            gpuTotalTimeMs += param.gpuTotalTimeMs;
            gpuApproxTimeMs += param.gpuApproxTimeMs;
        }
    }

    cout << "gpuSampleTimeMs: " << gpuSampleTimeMs / kNumTrials << endl;
    cout << "gpuFindThresholdTimeMs: " << gpuFindThresholdTimeMs / kNumTrials << endl;
    cout << "gpuCopyEligibleTimeMs: " << gpuCopyEligibleTimeMs / kNumTrials << endl;
    cout << "gpuRetreiveExactTimeMs: " << gpuRetreiveExactTimeMs / kNumTrials << endl;
    cout << "gpuTotalTimeMs: " << gpuTotalTimeMs / kNumTrials << endl;
    cout << "gpuApproxTimeMs: " << gpuApproxTimeMs / kNumTrials << endl;

    CHECK_CUDA(hipFree(param.d_score));
    CHECK_CUDA(hipHostFree(param.h_rstCpu));
    CHECK_CUDA(hipFree(param.d_rstGpu));
}

int main()
{
    runExp(4, 4000000, true, true);

    return 0;
}