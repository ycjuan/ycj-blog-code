#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <functional>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <cassert>
#include <numeric>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <string>
#include <omp.h>

#include "topk.cuh"
#include "common.cuh"
#include "util.cuh"

// Note: This is experimental code, so corner cases such as "numToRetrieve > numDocs" are not handled

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

void TopkSampling::malloc()
{
    size_t allocateInBytes;
    size_t totalAllocateInBytes = 0;

    allocateInBytes = kNumSamplesPerReq * kMaxNumReqs * sizeof(float);
    CHECK_CUDA(hipMalloc(&d_scoreSample, allocateInBytes));
    totalAllocateInBytes += allocateInBytes;
    cout << "allocated " << allocateInBytes / 1024.0 / 1024.0 / 1024.0 << " GiB for d_scoreSample" << endl;

    allocateInBytes = kMaxNumReqs * sizeof(float);
    CHECK_CUDA(hipMalloc(&d_scoreThreshold, allocateInBytes));
    totalAllocateInBytes += allocateInBytes;
    cout << "allocated " << allocateInBytes / 1024.0 / 1024.0 / 1024.0 << " GiB for d_scoreThreshold" << endl;

    allocateInBytes = kMaxNumReqs * kMaxEligiblePairsPerReq * sizeof(Pair);
    CHECK_CUDA(hipMalloc(&d_eligiblePairs, allocateInBytes));
    totalAllocateInBytes += allocateInBytes;
    cout << "allocated " << allocateInBytes / 1024.0 / 1024.0 / 1024.0 << " GiB for d_eligiblePairs" << endl;

    allocateInBytes = kMaxNumReqs * sizeof(int);
    CHECK_CUDA(hipMalloc(&d_copyCount, allocateInBytes));
    CHECK_CUDA(hipHostMalloc(&h_copyCount, allocateInBytes));
    totalAllocateInBytes += allocateInBytes;
    cout << "allocated " << allocateInBytes / 1024.0 / 1024.0 / 1024.0 << " GiB for d_copyCount" << endl;

    allocateInBytes = kMaxEligiblePairsPerReq * sizeof(Pair) + 10000000;
    thrustAllocator.malloc(allocateInBytes);
    totalAllocateInBytes += allocateInBytes;
    cout << "allocated " << allocateInBytes / 1024.0 / 1024.0 / 1024.0 << " GiB for thrustAllocator" << endl;

    cout << "total allocated " << totalAllocateInBytes / 1024.0 / 1024.0 / 1024.0 << " GiB" << endl;
}

void TopkSampling::free()
{
    CHECK_CUDA(hipFree(d_scoreSample));
    CHECK_CUDA(hipFree(d_scoreThreshold));
    CHECK_CUDA(hipFree(d_eligiblePairs));
    CHECK_CUDA(hipFree(d_copyCount));
    CHECK_CUDA(hipHostFree(h_copyCount));
    thrustAllocator.free();
}

void TopkSampling::retrieveTopk(TopkParam &param)
{
    CudaTimer timerTotal;
    CudaTimer timerApprox;
    timerTotal.tic();
    timerApprox.tic();

    // Step1 - Sample
    sample(param);

    // Step2 - Find threshold
    findThreshold(param);

    // Step3 - Copy eligible
    copyEligible(param);
    param.gpuApproxTimeMs = timerApprox.tocMs();

    // Step4 - retreiveExact
    retrieveExact(param);

    param.gpuTotalTimeMs = timerTotal.tocMs();
}

__global__ void sampleKernelNonRandom(TopkParam topkParam, float *d_scoreSample, size_t sampleSizePerReq)
{
    int wid = threadIdx.x + blockIdx.x * blockDim.x;

    if (wid < topkParam.numReqs * sampleSizePerReq)
    {
        int reqIdx = wid / sampleSizePerReq;
        int docIdx = wid % sampleSizePerReq;
        d_scoreSample[getMemAddr(reqIdx, docIdx, sampleSizePerReq)] = topkParam.d_score[getMemAddr(reqIdx, docIdx, topkParam.numDocs)];
    }
}

__global__ void sampleKernelRandom(TopkParam topkParam, float *d_scoreSample, size_t sampleSizePerReq, int docIdxGap)
{
    int wid = threadIdx.x + blockIdx.x * blockDim.x;

    if (wid < topkParam.numReqs * sampleSizePerReq)
    {
        int reqIdx = wid / sampleSizePerReq;
        int docIdxSample = wid % sampleSizePerReq;
        int docIdx = (docIdxSample * docIdxGap) % topkParam.numDocs;
        d_scoreSample[getMemAddr(reqIdx, docIdxSample, sampleSizePerReq)] = topkParam.d_score[getMemAddr(reqIdx, docIdx, topkParam.numDocs)];
    }
}

void TopkSampling::sample(TopkParam &param)
{
    CudaTimer timer;
    timer.tic();

    int blockSize = 256;
    int gridSize = (param.numReqs * kNumSamplesPerReq + blockSize - 1) / blockSize;
    if (param.useRandomSampling)
    {
        int docIdxGap = ceil((double)param.numDocs / kNumSamplesPerReq);
        // We sample every docIdxGap docs
        sampleKernelRandom<<<gridSize, blockSize>>>(param, d_scoreSample, kNumSamplesPerReq, docIdxGap);
    }
    else
    {
        sampleKernelNonRandom<<<gridSize, blockSize>>>(param, d_scoreSample, kNumSamplesPerReq);
    }
    CHECK_CUDA(hipDeviceSynchronize());

    param.gpuSampleTimeMs = timer.tocMs();
}

__global__ void updateThreshold(float *d_scoreSample, float *d_scoreThreshold, int numReqs, int thIdx, size_t kNumSamplesPerReq)
{
    int reqIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (reqIdx < numReqs)
    {
        d_scoreThreshold[reqIdx] = d_scoreSample[reqIdx * kNumSamplesPerReq + thIdx];
    }
}

void TopkSampling::findThreshold(TopkParam &param)
{
    CudaTimer timer;
    timer.tic();

    int thIdx = ceil((double)param.numToRetrieve / param.numDocs * kNumSamplesPerReq * 8);
    //omp_set_num_threads(4);
    //#pragma omp parallel for
    // mutlithreading does not help much here
    for (size_t reqIdx = 0; reqIdx < param.numReqs; reqIdx++)
    {
        thrust::sort(thrust::cuda::par(thrustAllocator),
                     d_scoreSample + reqIdx       * kNumSamplesPerReq,
                     d_scoreSample + (reqIdx + 1) * kNumSamplesPerReq,
                     thrust::greater<float>());
    }

    int blockSize = 256;
    int gridSize = (param.numReqs + blockSize - 1) / blockSize;
    updateThreshold<<<gridSize, blockSize>>>(d_scoreSample, d_scoreThreshold, param.numReqs, thIdx, kNumSamplesPerReq);

    param.gpuFindThresholdTimeMs = timer.tocMs();
}

__global__ void copyEligibleKernel(float *d_score,
                                   float *d_scoreThreshold,
                                   Pair *d_eligiblePairs,
                                   int *d_copyCount,
                                   int numReqs,
                                   int numDocs,
                                   size_t kMaxEligiblePairsPerReq)
{
    size_t wid = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (wid < numReqs * numDocs)
    {
        int reqIdx = wid / numDocs;
        int docIdx = wid % numDocs;
        size_t memAddr = getMemAddr(reqIdx, docIdx, numDocs);
        float score = d_score[memAddr];
        float threshold = d_scoreThreshold[reqIdx];
        if (score >= threshold)
        {
            int count = atomicAdd(d_copyCount + reqIdx, 1);
            if (count < kMaxEligiblePairsPerReq)
            {
                Pair pair;
                pair.reqIdx = reqIdx;
                pair.docIdx = docIdx;
                pair.score = score;
                d_eligiblePairs[reqIdx * kMaxEligiblePairsPerReq + count] = pair;
            }
        }
    }
}

void TopkSampling::copyEligible(TopkParam &param)
{
    /*
    for (size_t reqIdx = 0; reqIdx < param.numReqs; reqIdx++)
    {
        thrust::copy_if(thrust::device,
                        param.d_score + reqIdx * param.numDocs,
                        param.d_score + (reqIdx + 1) * param.numDocs,
                        param.dm_rst + reqIdx * param.numToRetrieve,
                        [thIdx = d_scoreThreshold[reqIdx]] __device__(float score) mutable {
                            return score >= thIdx;
                        });
    }
    */

    CudaTimer timer;
    timer.tic();

    CHECK_CUDA(hipMemset(d_copyCount, 0, param.numReqs * sizeof(int)));
    int blockSize = 256;
    int gridSize = (size_t(param.numReqs) * param.numDocs + blockSize - 1) / blockSize;
    copyEligibleKernel<<<gridSize, blockSize>>>(param.d_score,
                                                d_scoreThreshold,
                                                d_eligiblePairs,
                                                d_copyCount,
                                                param.numReqs,
                                                param.numDocs,
                                                kMaxEligiblePairsPerReq);

    CHECK_CUDA(hipDeviceSynchronize());

    param.gpuCopyEligibleTimeMs = timer.tocMs();
}

void TopkSampling::retrieveExact(TopkParam &param)
{
    CudaTimer timer;
    timer.tic();
    CHECK_CUDA(hipMemcpy(h_copyCount, d_copyCount, param.numReqs * sizeof(int), hipMemcpyDeviceToHost));
    for (size_t reqIdx = 0; reqIdx < param.numReqs; reqIdx++)
    {
        //TODO: dm -> d
        Pair *dm_eligiblePairsStart = d_eligiblePairs + reqIdx * kMaxEligiblePairsPerReq;
        Pair *dm_eligiblePairsEnd = dm_eligiblePairsStart + h_copyCount[reqIdx];
        thrust::stable_sort(thrust::cuda::par(thrustAllocator),
                            dm_eligiblePairsStart,
                            dm_eligiblePairsEnd,
                            ScorePredicator());
        thrust::copy(thrust::cuda::par(thrustAllocator),
                     dm_eligiblePairsStart,
                     dm_eligiblePairsStart + param.numToRetrieve,
                     param.d_rstGpu + reqIdx * param.numToRetrieve);
    }
    param.gpuRetreiveExactTimeMs = timer.tocMs();
}