#include <random>
#include "quant.cuh"

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

void QuantData::initRand(int numDocs, int numReqs, int numInt32)
{
    this->numDocs = numDocs;
    this->numReqs = numReqs;
    this->numInt32 = numInt32;
    this->docMemLayout = docMemLayout;
    this->reqMemLayout = reqMemLayout;

    CHECK_CUDA(hipMalloc(&d_doc, (size_t)numDocs * numInt32 * sizeof(T_QUANT)));
    CHECK_CUDA(hipMalloc(&d_req, (size_t)numReqs * numInt32 * sizeof(T_QUANT)));
    CHECK_CUDA(hipMalloc(&d_rstGpu, (size_t)numDocs * numReqs * sizeof(T_QUANT_RST)));
    CHECK_CUDA(hipHostMalloc(&h_doc, (size_t)numDocs * numInt32 * sizeof(T_QUANT)));
    CHECK_CUDA(hipHostMalloc(&h_req, (size_t)numReqs * numInt32 * sizeof(T_QUANT)));
    CHECK_CUDA(hipHostMalloc(&h_rstCpu, (size_t)numDocs * numReqs * sizeof(T_QUANT_RST)));

    T_QUANT uid = 0;
    for (int i = 0; i < numDocs; i++)
        for (int k = 0; k < numInt32; k++)
            h_doc[getMemAddr(i, k, numDocs, numInt32, docMemLayout)] = uid++;

    uid = 0;
    for (int j = 0; j < numReqs; j++)
    {
        for (int k = 0; k < numInt32; k++)
        {
            size_t addr = getMemAddr(j, k, numReqs, numInt32, reqMemLayout);
            h_req[addr] = uid++;
        }
    }

    CHECK_CUDA(hipMemcpy(d_doc, h_doc, (size_t)numDocs * numInt32 * sizeof(T_QUANT), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_req, h_req, (size_t)numReqs * numInt32 * sizeof(T_QUANT), hipMemcpyHostToDevice));
}

void QuantData::free()
{
    hipFree(d_doc);
    hipFree(d_req);
    hipFree(d_rstGpu);
    hipHostFree(h_doc);
    hipHostFree(h_req);
    hipHostFree(h_rstCpu);
}

void QuantData::print()
{
    ostringstream oss;
    oss << "numDocs: " << numDocs << ", numReqs: " << numReqs << ", numInt32: " << numInt32 << ", numBits: " << numInt32 * sizeof(T_QUANT) * 8 << endl;
    oss << "docMemLayout: " << (docMemLayout == ROW_MAJOR ? "ROW_MAJOR" : "COL_MAJOR") << endl;
    oss << "reqMemLayout: " << (reqMemLayout == ROW_MAJOR ? "ROW_MAJOR" : "COL_MAJOR") << endl;
    oss << "rstLayoutCpu: " << (rstLayoutCpu == ROW_MAJOR ? "ROW_MAJOR" : "COL_MAJOR") << endl;
    oss << "rstLayoutGpu: " << (rstLayoutGpu == ROW_MAJOR ? "ROW_MAJOR" : "COL_MAJOR") << endl;
    cout << oss.str();
}