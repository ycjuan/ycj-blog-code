#include "data.cuh"
#include "util.cuh"
#include <hip/hip_runtime.h>
#include <random>

namespace MatMatMulFromScratch
{

Data genData(int M, int N, int K)
{
    Data data;
    data.M = M;
    data.N = N;
    data.K = K;
    CHECK_CUDA(hipMallocManaged(&data.d_A, M * K * sizeof(T)));
    CHECK_CUDA(hipMallocManaged(&data.d_B, K * N * sizeof(T)));
    CHECK_CUDA(hipHostMalloc(&data.h_C, M * N * sizeof(float)));
    CHECK_CUDA(hipMallocManaged(&data.d_C, M * N * sizeof(float)));

    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(0.0, 1.0);
    for (int i = 0; i < data.M * data.K; i++)
    {
        data.d_A[i] = distribution(generator);
    }
    for (int i = 0; i < data.K * data.N; i++)
    {
        data.d_B[i] = distribution(generator);
    }

    return data;
}

void freeData(Data& data)
{
    if (data.d_A != nullptr)
    {
        hipFree(data.d_A);
    }
    if (data.d_B != nullptr)
    {
        hipFree(data.d_B);
    }
    if (data.h_C != nullptr)
    {
        hipHostFree(data.h_C);
    }
    if (data.d_C != nullptr)
    {
        hipFree(data.d_C);
    }
}

} // namespace BatchScalability