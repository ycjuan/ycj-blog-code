#include "data.cuh"
#include "util.cuh"
#include <hipblas.h>

namespace MatMatMulFromScratch
{
#define cublasErrCheck(stat)                         \
    {                                                \
        cublasErrCheck_((stat), __FILE__, __LINE__); \
    }
void cublasErrCheck_(hipblasStatus_t stat, const char* file, int line)
{
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

void methodCublas(Data& data)
{
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    float alpha = 1.0;
    float beta = 0.0;

    int MATRIX_M = data.M;
    int MATRIX_N = data.N;
    int MATRIX_K = data.K;

    T* a_fp16 = data.d_A;
    T* b_fp16 = data.d_B;
    float* c_cublas = data.d_C;

    //hipblasOperation_t trana = (kAIsRowMajor) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    //hipblasOperation_t tranb = (kBIsRowMajor) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    //int lda = (kAIsRowMajor) ? data.M : data.K;
    //int ldb = (kBIsRowMajor) ? data.K : data.N;
    //hipDataType aType = (std::is_same<T, half>::value) ? HIP_R_16F : HIP_R_16BF;
    //hipDataType bType = (std::is_same<T, half>::value) ? HIP_R_16F : HIP_R_16BF;
    
    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        MATRIX_M, MATRIX_N, MATRIX_K, 
        &alpha,
        a_fp16, HIP_R_16F, MATRIX_M,
        b_fp16, HIP_R_16F, MATRIX_K,
        &beta, 
        c_cublas, HIP_R_32F, MATRIX_M,
        HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    hipblasDestroy(cublasHandle);
}

} // namespace MatMatMulFromScratch