#include "data.cuh"
#include "util.cuh"
#include <hipblas.h>

namespace MatMatMulFromScratch
{
#define cublasErrCheck(stat)                         \
    {                                                \
        cublasErrCheck_((stat), __FILE__, __LINE__); \
    }
void cublasErrCheck_(hipblasStatus_t stat, const char* file, int line)
{
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

void methodCublas(Data& data)
{
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    float alpha = 1.0;
    float beta = 0.0;

    hipblasOperation_t trana = (kAIsRowMajor) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    //hipblasOperation_t tranb = (kBIsRowMajor) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    int lda = (kAIsRowMajor) ? data.K : data.M;
    //int ldb = (kBIsRowMajor) ? data.K : data.N;
    hipDataType aType = (std::is_same<T, half>::value) ? HIP_R_16F : HIP_R_16BF;
    hipDataType bType = (std::is_same<T, half>::value) ? HIP_R_16F : HIP_R_16BF;
    
    cublasErrCheck(hipblasGemmEx(cublasHandle, trana, HIPBLAS_OP_N, 
        data.M, data.N, data.K, 
        &alpha,
        data.d_A, aType, lda,
        data.d_B, bType, data.K,
        &beta, 
        data.d_C, HIP_R_32F, data.M,
        HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    hipblasDestroy(cublasHandle);
}

} // namespace MatMatMulFromScratch