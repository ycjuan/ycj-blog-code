#include "data.cuh"
#include "util.cuh"
#include <hipblas.h>

namespace MatMatMulFromScratch
{
#define cublasErrCheck(stat)                         \
    {                                                \
        cublasErrCheck_((stat), __FILE__, __LINE__); \
    }
void cublasErrCheck_(hipblasStatus_t stat, const char* file, int line)
{
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

void matMulCublas(Data& data)
{
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    float alpha = 1.0;
    float beta = 0.0;

    T* a_fp16 = data.d_A;
    T* b_fp16 = data.d_B;
    float* c_cublas = data.d_C;

    hipblasOperation_t trana = (kAIsRowMajor) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t tranb = (kBIsRowMajor) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    int lda = (kAIsRowMajor) ? data.K : data.M;
    int ldb = (kBIsRowMajor) ? data.N : data.K;
    hipDataType aType = (std::is_same<T, half>::value) ? HIP_R_16F : HIP_R_16BF;
    hipDataType bType = (std::is_same<T, half>::value) ? HIP_R_16F : HIP_R_16BF;

    cublasErrCheck(hipblasGemmEx(cublasHandle, trana, tranb,
        data.M, data.N, data.K,
        &alpha,
        a_fp16, aType, lda,
        b_fp16, bType, ldb,
        &beta,
        c_cublas, HIP_R_32F, data.M,
        HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    hipblasDestroy(cublasHandle);
}

} // namespace MatMatMulFromScratch