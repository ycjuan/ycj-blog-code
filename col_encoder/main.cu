#include "hip/hip_runtime.h"
#include <sstream>
#include <iostream>

#include "col_enc_gpu.cuh"
#include "col_enc_cpu.cuh"
#include "data_struct.cuh"
#include "data_cvt.cuh"
#include "data_gen.cuh"
#include "util.cuh"

using namespace std;

void compareResults(const std::vector<ScoringTask>& cpuTasks, const ScoringTasksGpu& gpuTasksRaw)
{
    using namespace std;

    // Copy results back from GPU
    vector<ScoringTask> gpuTasks = convertScoringTasksBackToCpu(gpuTasksRaw);

    // Compare CPU and GPU results
    for (size_t i = 0; i < cpuTasks.size(); ++i) 
    {
        float cpuResult = cpuTasks[i].result;
        float gpuResult = gpuTasks[i].result;
        float relativeError = abs(cpuResult - gpuResult) / (abs(cpuResult) + 1e-6f); // Avoid division by zero
        if (relativeError > 1e-3) // Use relative error for comparison
        {
            ostringstream oss;
            oss << "Mismatch at task " << i << ": CPU result = " << cpuTasks[i].result 
                << ", GPU result = " << gpuTasks[i].result;
            throw runtime_error(oss.str());
        }
    }
}

void runTest(const int kNumReqs, const int kNumDocs, const int kNumFields, const int kEmbDimPerField, const int kNumToScore, const int kNumTrials)
{
    using namespace std;

    // -------------------
    // Print test parameters
    cout << "kNumReqs: " << kNumReqs << ", "
         << "kNumDocs: " << kNumDocs << ", "
         << "kNumFields: " << kNumFields << ", "
         << "kEmbDimPerField: " << kEmbDimPerField << ", "
         << "kNumToScore: " << kNumToScore << endl;

    // -------------------
    // Random data CPU
    auto reqDataCpu = genRandColEncData(kNumReqs, kNumFields, kEmbDimPerField);
    auto docDataCpu = genRandColEncData(kNumDocs, kNumFields, kEmbDimPerField);
    auto taskDataCpu = genRandScoringTasks(kNumReqs, kNumToScore, kNumDocs);

    // -------------------
    // Convert to GPU data
    auto reqDataGpu = convertColEncDataToGpu(reqDataCpu);
    auto docDataGpu = convertColEncDataToGpu(docDataCpu);
    auto taskDataGpu = convertScoringTasksToGpu(taskDataCpu);

    // Malloc buffer
    EMB_T* d_tmpDocData = nullptr;
    size_t size_byte_tmpDocData = (size_t)kNumToScore * docDataGpu.numFields * docDataGpu.embDimPerField * sizeof(EMB_T);
    hipError_t hipError_t = hipMalloc(&d_tmpDocData, size_byte_tmpDocData);
    if (hipError_t != hipSuccess)
    {
        throw runtime_error("Failed to allocate device memory for d_tmpDocData: " + std::to_string(hipError_t));
    }

    float *d_tmpRst = nullptr;
    size_t size_byte_tmpRst = (size_t)kNumToScore * docDataGpu.numFields * sizeof(float);
    hipError_t = hipMalloc(&d_tmpRst, size_byte_tmpRst);
    if (hipError_t != hipSuccess)
    {
        throw runtime_error("Failed to allocate device memory for d_tmpRst: " + std::to_string(hipError_t));
    }

    // -------------------
    // Run scoring
    colEncScorerCpu(reqDataCpu, docDataCpu, taskDataCpu);
    // Create cuBLAS handle
    hipblasHandle_t cublasHandle;
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) 
    {
        throw runtime_error("Failed to create cuBLAS handle");
    }
    colEncScorerGpu(reqDataGpu, docDataGpu, taskDataGpu, 0, cublasHandle, d_tmpDocData, d_tmpRst);

    // -------------------
    // Compare results
    compareResults(taskDataCpu, taskDataGpu);

    // -------------------
    // Test latency
    Timer timer;
    for (int trial = -3; trial < kNumTrials; ++trial) 
    {
        if (trial == 0) 
        {
            timer.tic();
        }
        colEncScorerGpu(reqDataGpu, docDataGpu, taskDataGpu, 0, cublasHandle, d_tmpDocData, d_tmpRst);
    }
    float latencyMs = timer.tocMs() / kNumTrials;
    cout << "Average latency per trial: " << latencyMs << " ms" << endl;

    // -------------------
    // Compare results just in case
    compareResults(taskDataCpu, taskDataGpu);

    // -------------------
    // Free GPU data
    reqDataGpu.free();
    docDataGpu.free();
    taskDataGpu.free();
    hipFree(d_tmpDocData);
    hipFree(d_tmpRst);
    hipblasStatus_t = hipblasDestroy(cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) 
    {
        throw runtime_error("Failed to destroy cuBLAS handle: " + std::to_string(hipblasStatus_t));
    }
}

int main() 
{
    const int kNumReqs = 16;
    const int kNumDocs = 50000;
    const int kNumFields = 10;
    const int kEmbDimPerField = 512;
    const int kNumToScore = 1000;

    runTest(kNumReqs, kNumDocs, kNumFields, kEmbDimPerField, kNumToScore, 20);

    return 0;
}