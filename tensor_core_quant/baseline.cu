#include "hip/hip_runtime.h"
#include <bitset>

#include "common.cuh"
#include "util.cuh"

void matMulCpu(Data data, Setting setting)
{
    Timer timer;
    timer.tic();
    #pragma omp parallel for
    for (int i = 0; i < data.numDocs; i++)
    {
        for (int j = 0; j < data.numReqs; j++)
        {
            uint16_t totalCount = 0;
            for (int k = 0; k < data.numInt64; k++)
            {
                uint64_t reqVal = data.d_req[getMemAddr(j, k, data.numReqs, data.numInt64, data.reqMemLayout)];
                uint64_t docVal = data.d_doc[getMemAddr(i, k, data.numDocs, data.numInt64, data.docMemLayout)];
                uint64_t bitwiseRst = ~ (reqVal ^ docVal);
                bitset<64> bits(bitwiseRst);
                totalCount += bits.count();
            }
            data.h_rst_cpu[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutCpu)] = totalCount;
        }
    }
    cout << "CPU time: " << timer.tocMs() << " ms" << endl;
}

__global__ void matMul(Data data)
{
    int threadId = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    int i = threadId / data.numReqs;
    int j = threadId % data.numReqs;

    if (i < data.numDocs && j < data.numReqs)
    {
        uint16_t totalCount = 0;
        for (int k = 0; k < data.numInt64; k++)
        {
            uint64_t reqVal = data.d_req[getMemAddr(j, k, data.numReqs, data.numInt64, data.reqMemLayout)];
            uint64_t docVal = data.d_doc[getMemAddr(i, k, data.numDocs, data.numInt64, data.docMemLayout)];            
            uint64_t bitwiseRst = ~ (reqVal ^ docVal);
            totalCount += __popcll(bitwiseRst); // This counts the number of "1" in the 64bit bitwiseAnd
        }
        data.d_rst_kernel[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutGpuKernel)] = totalCount;
    }
}

void matMulKernel(Data data, Setting setting)
{
    int blockSize = 512;
    int gridSize = size_t(data.numDocs) * data.numReqs / blockSize;
    CudaTimer timer;
    for (int t = -3; t < setting.kNumTrials; t++)
    {
        if (t == 0)
            timer.tic();
        matMul<<<gridSize, blockSize>>>(data);
        hipDeviceSynchronize();
        hipError_t status = hipGetLastError();
        if (status != hipSuccess)
        {
            ostringstream oss;
            oss << "Kernel launch failed with error: " << hipGetErrorString(status) << "\n";
            throw runtime_error(oss.str());
        }
    }
    cout << "Kernel time: " << timer.tocMs() / setting.kNumTrials << " ms" << endl;
}
