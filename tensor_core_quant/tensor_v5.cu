#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
This file is modified from: 

https://github.com/NVIDIA-developer-blog/code-samples/blob/708ce9137eb5ac7682f788e5d5b8279c7e2578ed/posts/tensor-cores/simpleTensorCoreGEMM.cu

*/

#include <stdio.h>
#include <hiprand.h>
#include <hipblas.h>

#include "common.cuh"
#include "util.cuh"

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}


#include <mma.h>
using namespace nvcuda;

// The only dimensions currently supported by WMMA
const int WMMA_M = 8;
const int WMMA_N = 8;
const int WMMA_K = 128;


// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16. 
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.
__global__ void wmma_example(T1 *A, T1 *B, T2 *C, int M, int n, int k) {

   using namespace nvcuda::wmma::experimental;

   // Tile using a 2D grid
   int bx = blockIdx.x * blockDim.y + threadIdx.y;
   int by = blockIdx.y;

   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, precision::b1, wmma::row_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, precision::b1, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, T2> c_frag;

   wmma::fill_fragment(c_frag, 0);

   for (int j = 0; j < (k / 128); j++)
   {
      load_matrix_sync(a_frag, A + bx * 8 * k / 32 + j * 128 * 8 / 32, 128);
      load_matrix_sync(b_frag, B + by * 8 * k / 32 + j * 128 * 8 / 32, 128);
      bmma_sync(c_frag, a_frag, b_frag, c_frag, bmmaBitOpXOR, bmmaAccumulateOpPOPC);
   }

   store_matrix_sync(C + (bx * 8 * n + by * 8), c_frag, n, wmma::mem_row_major);
   /*
   if (bx == 0 && by == 0)
   {
      printf("a_frag.x = %u, b_frag.x = %u, c_frag.x = %u\n", a_frag.x, b_frag.x, c_frag.x);
      printf("A[0] = %u, B[0] = %u, C[0] = %d\n", A[0], B[0], C[0]);
   }
   */
}

void quantWMMA(Data data, Setting setting) {

   int MATRIX_M = data.numDocs;
   int MATRIX_N = data.numReqs;
   int MATRIX_K = data.numT1;

   T1 *a_fp16 = data.d_doc;
   T1 *b_fp16 = data.d_req;

   T2 *c_wmma = data.d_rst_wmma;

   printf("\nM = %d, N = %d, K = %d.\n\n", MATRIX_M, MATRIX_N, MATRIX_K);
   
   // First: using WMMA
   dim3 blockDim(32, 2);
   dim3 gridDim(MATRIX_M/16, MATRIX_N/8);

   cout << "blockDim: " << blockDim.x << " " << blockDim.y << endl;
   cout << "gridDim: " << gridDim.x << " " << gridDim.y << endl;
 
   printf("Running with wmma...\n");
   CudaTimer timer;
   for (int t = -3; t < setting.kNumTrials; t++)
   {
      if (t == 0)
         timer.tic();
      wmma_example <<< gridDim, blockDim >>> (a_fp16, b_fp16, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K);
      cudaErrCheck(hipDeviceSynchronize());
      cudaErrCheck(hipGetLastError());
   }
   cout << "wmma took " << timer.tocMs() / setting.kNumTrials << "ms" << endl;
}


