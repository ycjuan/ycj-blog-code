#include "hip/hip_runtime.h"
/*
This file is modified from: 

https://github.com/NVIDIA-developer-blog/code-samples/blob/708ce9137eb5ac7682f788e5d5b8279c7e2578ed/posts/tensor-cores/simpleTensorCoreGEMM.cu

https://github.com/pnnl/TCBNN/blob/de4713445fd1cd772ad176080a0ff61a5f862e3b/bmm/tensorcore_kernel.cu#L336

Thanks to the authors of the original code!
*/

#include <stdio.h>
#include <hiprand.h>
#include <hipblas.h>

#include "common.cuh"
#include "util.cuh"

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

#include <mma.h>
using namespace nvcuda;

const int WMMA_M = 8;
const int WMMA_N = 8;
const int WMMA_K = 128;

__global__ void quantWmmaUnrollKernelV2(const unsigned *a, const unsigned *b, int *c, const unsigned M, const unsigned N, const unsigned K)
{
   using namespace nvcuda::wmma::experimental;
   size_t lda = K;
   size_t ldb = K;
   size_t ldc = N;

   // Tile using a 2D grid
   size_t warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   size_t warpN = (blockIdx.y * blockDim.y + threadIdx.y);

   // 8: WWMA_M, 8: WMMA_N, 128: WMMA_K
   wmma::fragment<wmma::matrix_a, 8, 8, 128, precision::b1, wmma::row_major> a_frag0;
   wmma::fragment<wmma::matrix_a, 8, 8, 128, precision::b1, wmma::row_major> a_frag1;
   wmma::fragment<wmma::matrix_b, 8, 8, 128, precision::b1, wmma::col_major> b_frag0;
   wmma::fragment<wmma::matrix_b, 8, 8, 128, precision::b1, wmma::col_major> b_frag1;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag00;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag01;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag10;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag11;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag20;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag21;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag30;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag31;
   wmma::fill_fragment(c_frag00, 0);
   wmma::fill_fragment(c_frag01, 0);
   wmma::fill_fragment(c_frag10, 0);
   wmma::fill_fragment(c_frag11, 0);
   wmma::fill_fragment(c_frag20, 0);
   wmma::fill_fragment(c_frag21, 0);
   wmma::fill_fragment(c_frag30, 0);
   wmma::fill_fragment(c_frag31, 0);

   size_t lda32 = lda / 32; // 32: number of bytes of int32
   size_t ldb32 = ldb / 32; // 32: number of bytes of int32
   for (int i = 0; i < K; i += WMMA_K) {
      int i32 = i / 32; // 32: number of bytes of int32
      size_t aRow0 = warpM * WMMA_M * 2; // 4: because we do 4x unrolling
      size_t aCol0 = i32;

      size_t aRow1 = aRow0 + WMMA_M;
      size_t aCol1 = i32;

      size_t bRow0 = i32;
      size_t bCol0 = warpN * WMMA_N * 2; // 4: because we do 4x unrolling

      size_t bRow1 = i32;
      size_t bCol1 = bCol0 + WMMA_N;

      wmma::load_matrix_sync(a_frag0, a + aRow0 * lda32 + aCol0, lda);
      wmma::load_matrix_sync(a_frag1, a + aRow1 * lda32 + aCol1, lda);
      wmma::load_matrix_sync(b_frag0, b + bCol0 * ldb32 + bRow0, ldb);
      wmma::load_matrix_sync(b_frag1, b + bCol1 * ldb32 + bRow1, ldb);

      // Perform the matrix multiplication
      wmma::bmma_sync(c_frag00, a_frag0, b_frag0, c_frag00);
      wmma::bmma_sync(c_frag01, a_frag0, b_frag1, c_frag01);
      wmma::bmma_sync(c_frag10, a_frag1, b_frag0, c_frag10);
      wmma::bmma_sync(c_frag11, a_frag1, b_frag1, c_frag11);

   }

   int cRow0 = warpM * WMMA_M * 2; // 4: because we do 4x unrolling
   int cRow1 = cRow0 + WMMA_M;

   int cCol0 = warpN * WMMA_N * 2; // 4: because we do 4x unrolling
   int cCol1 = cCol0 + WMMA_N;

#pragma unroll
   for (int i = 0; i < c_frag00.num_elements; i++)
   {
      c_frag00.x[i] = K - c_frag00.x[i];
      c_frag01.x[i] = K - c_frag01.x[i];
      c_frag10.x[i] = K - c_frag10.x[i];
      c_frag11.x[i] = K - c_frag11.x[i];
      c_frag20.x[i] = K - c_frag20.x[i];
      c_frag21.x[i] = K - c_frag21.x[i];
      c_frag30.x[i] = K - c_frag30.x[i];
      c_frag31.x[i] = K - c_frag31.x[i];
   }

   wmma::store_matrix_sync(c + cRow0 * ldc + cCol0, c_frag00, ldc, wmma::mem_row_major);
   wmma::store_matrix_sync(c + cRow0 * ldc + cCol1, c_frag01, ldc, wmma::mem_row_major);
   wmma::store_matrix_sync(c + cRow1 * ldc + cCol0, c_frag10, ldc, wmma::mem_row_major);
   wmma::store_matrix_sync(c + cRow1 * ldc + cCol1, c_frag11, ldc, wmma::mem_row_major);
   
}

void quantWmmaUnrollV2(Data data, Setting setting) {

   int MATRIX_M = data.numDocs;
   int MATRIX_N = data.numReqs;
   int MATRIX_K = data.numInt;

   T_QUANT *a = data.d_doc;
   T_QUANT *b = data.d_req;

   T_RST *c_wmma = data.d_rstWmmaUnroll;

   
   dim3 gridDim;
   dim3 blockDim;
 
   blockDim.x = 512;
   blockDim.y = 1;

   gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32) / 2; // 32: warpSize; 4: because we do 4x unrolling
   gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y) / 2; // 4: because we do 4x unrolling

   printf("\nRunning with wmma (unroll)...\n");
   printf("M = %d, N = %d, K = %d.\n", MATRIX_M, MATRIX_N, MATRIX_K);
   cout << "blockDim: " << blockDim.x << " " << blockDim.y << endl;
   cout << "gridDim: " << gridDim.x << " " << gridDim.y << endl;
   CudaTimer timer;
   for (int t = -3; t < setting.kNumTrials; t++)
   {
      if (t == 0)
         timer.tic();
      quantWmmaUnrollKernelV2 <<< gridDim, blockDim >>> (a, b, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K * 32); // 32: number of bytes of int32
      CHECK_CUDA(hipDeviceSynchronize());
      CHECK_CUDA(hipGetLastError());
   }
   cout << "wmma (unroll) took " << timer.tocMs() / setting.kNumTrials << "ms" << endl;
}


