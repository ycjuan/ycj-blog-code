#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
This file is modified from: 

https://github.com/NVIDIA-developer-blog/code-samples/blob/708ce9137eb5ac7682f788e5d5b8279c7e2578ed/posts/tensor-cores/simpleTensorCoreGEMM.cu

https://github.com/pnnl/TCBNN/blob/de4713445fd1cd772ad176080a0ff61a5f862e3b/bmm/tensorcore_kernel.cu#L336
*/

#include <stdio.h>
#include <hiprand.h>
#include <hipblas.h>

#include "common.cuh"
#include "util.cuh"

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#include <mma.h>
using namespace nvcuda;

__global__ void wmma_example(const unsigned *A, const unsigned *B, int *C, const unsigned m, const unsigned n, const unsigned k)
{
   using namespace nvcuda::wmma::experimental;
   int bx = blockIdx.x * blockDim.y + threadIdx.y;
   int by = blockIdx.y;

   wmma::fragment<wmma::matrix_a, 8, 8, 128, precision::b1, wmma::row_major> a_frag;
   wmma::fragment<wmma::matrix_b, 8, 8, 128, precision::b1, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag;
   wmma::fill_fragment(c_frag, 0);

   for (int j = 0; j < (k / 128); j++)
   {
      load_matrix_sync(a_frag, A + bx * 8 * k / 32 + j * 128 * 8 / 32, 128);
      load_matrix_sync(b_frag, B + by * 8 * k / 32 + j * 128 * 8 / 32, 128);

      bmma_sync(c_frag, a_frag, b_frag, c_frag, bmmaBitOpXOR, bmmaAccumulateOpPOPC);
   }

#pragma unroll
   for (int i = 0; i < c_frag.num_elements; i++)
      c_frag.x[i] = k - c_frag.x[i];

   store_matrix_sync(C + (bx * 8 * n + by * 8), c_frag, n, wmma::mem_row_major);
}

void quantWMMA(Data data, Setting setting) {

   int MATRIX_M = data.numDocs;
   int MATRIX_N = data.numReqs;
   int MATRIX_K = data.numT1;

   T1 *a_fp16 = data.d_doc;
   T1 *b_fp16 = data.d_req;

   T2 *c_wmma = data.d_rst_wmma;

   printf("\nM = %d, N = %d, K = %d.\n\n", MATRIX_M, MATRIX_N, MATRIX_K);
   
   // First: using WMMA
   dim3 blockDim(32, 2);
   dim3 gridDim(MATRIX_M/16, MATRIX_N/8);

   cout << "blockDim: " << blockDim.x << " " << blockDim.y << endl;
   cout << "gridDim: " << gridDim.x << " " << gridDim.y << endl;

   printf("Running with wmma...\n");
   CudaTimer timer;
   for (int t = -3; t < setting.kNumTrials; t++)
   {
      if (t == 0)
         timer.tic();
      wmma_example <<< gridDim, blockDim >>> (a_fp16, b_fp16, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K * 32);
      cudaErrCheck(hipDeviceSynchronize());
      cudaErrCheck(hipGetLastError());
   }
   cout << "wmma took " << timer.tocMs() / setting.kNumTrials << "ms" << endl;
}


