#include "hip/hip_runtime.h"
/*
This file is modified from: 

https://github.com/NVIDIA-developer-blog/code-samples/blob/708ce9137eb5ac7682f788e5d5b8279c7e2578ed/posts/tensor-cores/simpleTensorCoreGEMM.cu

https://github.com/pnnl/TCBNN/blob/de4713445fd1cd772ad176080a0ff61a5f862e3b/bmm/tensorcore_kernel.cu#L336
*/

#include <stdio.h>
#include <hiprand.h>
#include <hipblas.h>

#include "common.cuh"
#include "util.cuh"

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#include <mma.h>
using namespace nvcuda;

const int WMMA_M = 8;
const int WMMA_N = 8;
const int WMMA_K = 128;

__global__ void quantWmmaKernel(const unsigned *a, const unsigned *b, int *c, const unsigned M, const unsigned N, const unsigned K)
{
   using namespace nvcuda::wmma::experimental;
   int lda = K;
   int ldb = K;
   int ldc = N;

   // Tile using a 2D grid
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

   wmma::fragment<wmma::matrix_a, 8, 8, 128, precision::b1, wmma::row_major> a_frag0;
   wmma::fragment<wmma::matrix_a, 8, 8, 128, precision::b1, wmma::row_major> a_frag1;
   wmma::fragment<wmma::matrix_b, 8, 8, 128, precision::b1, wmma::col_major> b_frag0;
   wmma::fragment<wmma::matrix_b, 8, 8, 128, precision::b1, wmma::col_major> b_frag1;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag00;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag01;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag10;
   wmma::fragment<wmma::accumulator, 8, 8, 128, int> c_frag11;
   wmma::fill_fragment(c_frag00, 0);
   wmma::fill_fragment(c_frag01, 0);
   wmma::fill_fragment(c_frag10, 0);
   wmma::fill_fragment(c_frag11, 0);

   int lda32 = lda / 32;
   int ldb32 = ldb / 32;
   for (int i = 0; i < K; i += WMMA_K) {
      int i32 = i / 32;
      size_t aRow0 = warpM * WMMA_M * 2;
      size_t aCol0 = i32;

      size_t aRow1 = aRow0 + WMMA_M;
      size_t aCol1 = i32;

      size_t bRow0 = i32;
      size_t bCol0 = warpN * WMMA_N * 2;

      size_t bRow1 = i32;
      size_t bCol1 = bCol0 + WMMA_N;

      wmma::load_matrix_sync(a_frag0, a + aRow0 * lda32 + aCol0, lda);
      wmma::load_matrix_sync(a_frag1, a + aRow1 * lda32 + aCol1, lda);
      wmma::load_matrix_sync(b_frag0, b + bCol0 * ldb32 + bRow0, ldb);
      wmma::load_matrix_sync(b_frag1, b + bCol1 * ldb32 + bRow1, ldb);

      // Perform the matrix multiplication
      wmma::bmma_sync(c_frag00, a_frag0, b_frag0, c_frag00);
      wmma::bmma_sync(c_frag01, a_frag0, b_frag1, c_frag01);
      wmma::bmma_sync(c_frag10, a_frag1, b_frag0, c_frag10);
      wmma::bmma_sync(c_frag11, a_frag1, b_frag1, c_frag11);
   }

   int cRow0 = warpM * WMMA_M * 2;
   int cCol0 = warpN * WMMA_N * 2;
   int cRow1 = cRow0 + WMMA_M;
   int cCol1 = cCol0 + WMMA_N;

#pragma unroll
   for (int i = 0; i < c_frag00.num_elements; i++)
   {
      c_frag00.x[i] = K - c_frag00.x[i];
      c_frag01.x[i] = K - c_frag01.x[i];
      c_frag10.x[i] = K - c_frag10.x[i];
      c_frag11.x[i] = K - c_frag11.x[i];
   }

   wmma::store_matrix_sync(c + cRow0 * ldc + cCol0, c_frag00, ldc, wmma::mem_row_major);
   wmma::store_matrix_sync(c + cRow0 * ldc + cCol1, c_frag01, ldc, wmma::mem_row_major);
   wmma::store_matrix_sync(c + cRow1 * ldc + cCol0, c_frag10, ldc, wmma::mem_row_major);
   wmma::store_matrix_sync(c + cRow1 * ldc + cCol1, c_frag11, ldc, wmma::mem_row_major);
   
}

void quantWMMA(Data data, Setting setting) {

   int MATRIX_M = data.numDocs;
   int MATRIX_N = data.numReqs;
   int MATRIX_K = data.numT1;

   T1 *a_fp16 = data.d_doc;
   T1 *b_fp16 = data.d_req;

   T2 *c_wmma = data.d_rst_wmma;

   printf("\nM = %d, N = %d, K = %d.\n\n", MATRIX_M, MATRIX_N, MATRIX_K);
   
   // First: using WMMA
   dim3 gridDim;
   dim3 blockDim;
 
   // blockDim.x must be a multple of warpSize
   // 128x4 means we have 16 warps and a block computes a 64x64 output tile
   blockDim.x = 128;
   blockDim.y = 4;

   gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32) / 2;
   gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y) / 2;

   cout << "blockDim: " << blockDim.x << " " << blockDim.y << endl;
   cout << "gridDim: " << gridDim.x << " " << gridDim.y << endl;

   printf("Running with wmma...\n");
   CudaTimer timer;
   for (int t = -3; t < setting.kNumTrials; t++)
   {
      if (t == 0)
         timer.tic();
      quantWmmaKernel <<< gridDim, blockDim >>> (a_fp16, b_fp16, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K * 32);
      cudaErrCheck(hipDeviceSynchronize());
      cudaErrCheck(hipGetLastError());
   }
   cout << "wmma took " << timer.tocMs() / setting.kNumTrials << "ms" << endl;
}


