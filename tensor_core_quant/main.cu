#include <string>
#include <stdexcept>
#include <iostream>
#include <random>
#include <sstream>
#include <hipblas.h>
#include <type_traits>

#include "util.cuh"
#include "common.cuh"
#include "methods.cuh"

using namespace std;

int kNumDocs = 1 << 18;
int kNumReqs = 1 << 7;
int kNumInt = 1 << 5;
int kNumTrials = 100;
MemLayout kMemLayoutDoc = ROW_MAJOR; 
// IMPORTANT: Don't change this. a_frag in WMMA requires ROW_MAJOR
MemLayout kMemLayoutReq = ROW_MAJOR; 
// IMPORTANT: Don't change this. b_frag in WMMA requires COL_MAJOR. 
// However, since the matrix here has a shape of (numReqs, numInt), setting ROW_MAJOR here is equivalent to COL_MAJOR of a (numInt, numReqs) matrix
MemLayout kMemLayoutRstCpu = ROW_MAJOR;
MemLayout kMemLayoutRstGpuCuda = ROW_MAJOR;
MemLayout kMemLayoutRstGpuTensorSimple = ROW_MAJOR;
MemLayout kMemLayoutRstGpuTensorUnroll = ROW_MAJOR;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

Data genData()
{
    Data data;
    data.numDocs = kNumDocs;
    data.numReqs = kNumReqs;
    data.numInt = kNumInt;
    data.docMemLayout = kMemLayoutDoc;
    data.reqMemLayout = kMemLayoutReq;
    data.rstLayoutCpu = kMemLayoutRstCpu;
    data.rstLayoutGpuCuda = kMemLayoutRstGpuCuda;
    data.rstLayoutGpuTensorSimple = kMemLayoutRstGpuTensorSimple;
    data.rstLayoutGpuTensorUnroll = kMemLayoutRstGpuTensorUnroll;
    data.print();
    
    CHECK_CUDA(hipMallocManaged(&data.d_doc, (size_t)data.numDocs * data.numInt * sizeof(T_QUANT)));
    CHECK_CUDA(hipMallocManaged(&data.d_req, (size_t)data.numReqs * data.numInt * sizeof(T_QUANT)));
    CHECK_CUDA(hipMallocManaged(&data.d_rst_kernel, (size_t)data.numDocs * data.numReqs * sizeof(T_RST)));
    CHECK_CUDA(hipMallocManaged(&data.d_rstTensorSimple, (size_t)data.numDocs * data.numReqs * sizeof(T_RST)));
    CHECK_CUDA(hipMallocManaged(&data.d_rstTensorUnroll, (size_t)data.numDocs * data.numReqs * sizeof(T_RST)));
    CHECK_CUDA(hipHostMalloc(&data.h_rst_cpu, (size_t)data.numDocs * data.numReqs * sizeof(T_RST)));


    default_random_engine generator;
    uniform_int_distribution<T_QUANT> distribution;

    T_QUANT uid = 0;
    for (int i = 0; i < data.numDocs; i++)
        for (int k = 0; k < data.numInt; k++)
            data.d_doc[getMemAddr(i, k, data.numDocs, data.numInt, data.docMemLayout)] = uid++;
    uid = 0;

    for (int j = 0; j < data.numReqs; j++)
    {
        for (int k = 0; k < data.numInt; k++)
        {
            size_t addr = getMemAddr(j, k, data.numReqs, data.numInt, data.reqMemLayout);
            data.d_req[addr] = uid++;
        }
    }
    return data;
}

void checkRst(Data data)
{
    for (int i = 0; i < data.numDocs; i++)
    {
        for (int j = 0; j < data.numReqs; j++)
        {
            T_RST cpuVal = data.h_rst_cpu[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutCpu)];
            T_RST gpuCudaVal = data.d_rst_kernel[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutGpuCuda)];
            T_RST gpuTensorSimpleVal = data.d_rstTensorSimple[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutGpuTensorSimple)];
            T_RST gpuTensorUnrollVal = data.d_rstTensorUnroll[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutGpuTensorUnroll)];

            if (cpuVal != gpuCudaVal)
            {
                cout << "Kernel error at (" << i << ", " << j << "): " << cpuVal << " != " << gpuCudaVal << endl;
                return;
            }
            
            if (cpuVal != gpuTensorSimpleVal)
            {
                cout << "Wmma error at (" << i << ", " << j << "): " << cpuVal << " != " << gpuTensorSimpleVal << endl;
                return;
            }

            if (cpuVal != gpuTensorUnrollVal)
            {
                cout << "Wmma unroll error at (" << i << ", " << j << "): " << cpuVal << " != " << gpuTensorUnrollVal << endl;
                return;
            }
        }
    }

    cout << "\nAll results are correct!" << endl;
}

int main()
{
    Data data = genData();
    Setting setting;
    setting.kNumTrials = kNumTrials;

    quantGpuCuda(data, setting);
    quantCpu(data, setting);
    quantWmmaSimple(data, setting);
    quantWmmaUnroll(data, setting);

    checkRst(data);

    data.free();

    return 0;
}
