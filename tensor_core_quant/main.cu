#include "hip/hip_runtime.h"
#include <string>
#include <stdexcept>
#include <iostream>
#include <random>
#include <sstream>
#include <hipblas.h>
#include <type_traits>
#include <bitset>

#include "util.cuh"

using namespace std;

enum MemLayout
{
    ROW_MAJOR,
    COL_MAJOR
};

int kNumDocs = 1 << 20;
int kNumReqs = 1 << 0;
int kNumInt64 = 1 << 10;
int kNumTrials = 100;
MemLayout kMemLayoutDoc = COL_MAJOR;
MemLayout kMemLayoutReq = ROW_MAJOR;
MemLayout kMemLayoutRstCpu = COL_MAJOR;
MemLayout kMemLayoutRstGpuKernel = COL_MAJOR;
MemLayout kMemLayoutRstGpuTensor = COL_MAJOR;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

__device__ __host__ size_t getMemAddr(int i, int j, int M, int N, MemLayout layout)
{
    if (layout == ROW_MAJOR)
        return (size_t)i * N + j;
    else
        return (size_t)j * M + i;
}

struct Data
{
    int numDocs;
    int numReqs;
    int numInt64;
    uint64_t *d_doc; // M=numDocs x N=numInt64
    uint64_t *d_req; // M=numReqs x N=numInt64
    uint16_t *d_rst_kernel; // M=numDocs x N=numReqs
    uint16_t *d_rst_cublas; // M=numDocs x N=numReqs
    uint16_t *h_rst_cpu;
    MemLayout docMemLayout;
    MemLayout reqMemLayout;
    MemLayout rstLayoutCpu;
    MemLayout rstLayoutGpuKernel;
    MemLayout rstLayoutGpuCublas;

    void free()
    {
        hipFree(d_doc);
        hipFree(d_req);
        hipFree(d_rst_kernel);
        hipFree(d_rst_cublas);
        hipHostFree(h_rst_cpu);
    }

    void print()
    {
        ostringstream oss;
        oss << "numDocs: " << numDocs << ", numReqs: " << numReqs << ", numInt64: " << numInt64 << endl;
        oss << "docMemLayout: " << (docMemLayout == ROW_MAJOR ? "ROW_MAJOR" : "COL_MAJOR") << endl;
        oss << "reqMemLayout: " << (reqMemLayout == ROW_MAJOR ? "ROW_MAJOR" : "COL_MAJOR") << endl;
        oss << "rstLayoutCpu: " << (rstLayoutCpu == ROW_MAJOR ? "ROW_MAJOR" : "COL_MAJOR") << endl;
        oss << "rstLayoutGpuKernel: " << (rstLayoutGpuKernel == ROW_MAJOR ? "ROW_MAJOR" : "COL_MAJOR") << endl;
        oss << "rstLayoutGpuCublas: " << (rstLayoutGpuCublas == ROW_MAJOR ? "ROW_MAJOR" : "COL_MAJOR") << endl;
        cout << oss.str();
    }
};

Data genData()
{
    Data data;
    data.numDocs = kNumDocs;
    data.numReqs = kNumReqs;
    data.numInt64 = kNumInt64;
    data.docMemLayout = kMemLayoutDoc;
    data.reqMemLayout = kMemLayoutReq;
    data.rstLayoutCpu = kMemLayoutRstCpu;
    data.rstLayoutGpuKernel = kMemLayoutRstGpuKernel;
    data.rstLayoutGpuCublas = kMemLayoutRstGpuTensor;
    data.print();
    
    CHECK_CUDA(hipMallocManaged(&data.d_doc, (size_t)data.numDocs * data.numInt64 * sizeof(uint64_t)));
    CHECK_CUDA(hipMallocManaged(&data.d_req, (size_t)data.numReqs * data.numInt64 * sizeof(uint64_t)));
    CHECK_CUDA(hipMallocManaged(&data.d_rst_kernel, (size_t)data.numDocs * data.numReqs * sizeof(uint16_t)));
    CHECK_CUDA(hipMallocManaged(&data.d_rst_cublas, (size_t)data.numDocs * data.numReqs * sizeof(uint16_t)));
    CHECK_CUDA(hipHostMalloc(&data.h_rst_cpu, (size_t)data.numDocs * data.numReqs * sizeof(uint16_t)));

    default_random_engine generator;
    uniform_int_distribution<uint64_t> distribution;
    for (int i = 0; i < data.numDocs * data.numInt64; i++)
        data.d_doc[i] = distribution(generator);
    for (int i = 0; i < data.numReqs * data.numInt64; i++)
        data.d_req[i] = distribution(generator);

    return data;
}

void checkData(Data data)
{
    for (int i = 0; i < data.numDocs; i++)
    {
        for (int j = 0; j < data.numReqs; j++)
        {
            uint16_t cpuVal = data.h_rst_cpu[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutCpu)];
            uint16_t gpuKernelVal = data.d_rst_kernel[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutGpuKernel)];
            uint16_t gpuCublasVal = data.d_rst_cublas[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutGpuCublas)];

            if (abs(cpuVal - gpuKernelVal) / abs(gpuKernelVal) > 1e-3)
            {
                cout << "Kernel error at (" << i << ", " << j << "): " << cpuVal << " != " << gpuKernelVal << endl;
                return;
            }
            
            if (abs(cpuVal - gpuCublasVal) / abs(gpuKernelVal) > 1e-3)
            {
                cout << "Cublas error at (" << i << ", " << j << "): " << cpuVal << " != " << gpuCublasVal << endl;
                return;
            }
        }
    }
}

void matMulCpu(Data data)
{
    Timer timer;
    timer.tic();
    #pragma omp parallel for
    for (int i = 0; i < data.numDocs; i++)
    {
        for (int j = 0; j < data.numReqs; j++)
        {
            uint16_t totalCount = 0;
            for (int k = 0; k < data.numInt64; k++)
            {
                uint64_t reqVal = data.d_req[getMemAddr(j, k, data.numReqs, data.numInt64, data.reqMemLayout)];
                uint64_t docVal = data.d_doc[getMemAddr(i, k, data.numDocs, data.numInt64, data.docMemLayout)];
                uint64_t bitwiseRst = ~ (reqVal ^ docVal);
                bitset<64> bits(bitwiseRst);
                totalCount += bits.count();
            }
            data.h_rst_cpu[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutCpu)] = totalCount;
        }
    }
    cout << "CPU time: " << timer.tocMs() << " ms" << endl;
}

__global__ void matMul(Data data)
{
    int threadId = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    int i = threadId / data.numReqs;
    int j = threadId % data.numReqs;

    if (i < data.numDocs && j < data.numReqs)
    {
        uint16_t totalCount = 0;
        for (int k = 0; k < data.numInt64; k++)
        {
            uint64_t reqVal = data.d_req[getMemAddr(j, k, data.numReqs, data.numInt64, data.reqMemLayout)];
            uint64_t docVal = data.d_doc[getMemAddr(i, k, data.numDocs, data.numInt64, data.docMemLayout)];            
            uint64_t bitwiseRst = ~ (reqVal ^ docVal);
            totalCount += __popcll(bitwiseRst); // This counts the number of "1" in the 64bit bitwiseAnd
        }
        data.d_rst_kernel[getMemAddr(i, j, data.numDocs, data.numReqs, data.rstLayoutGpuKernel)] = totalCount;
    }
}

void matMulKernel(Data data)
{
    int blockSize = 512;
    int gridSize = size_t(data.numDocs) * data.numReqs / blockSize;
    CudaTimer timer;
    for (int t = -3; t < kNumTrials; t++)
    {
        if (t == 0)
            timer.tic();
        matMul<<<gridSize, blockSize>>>(data);
        hipDeviceSynchronize();
        hipError_t status = hipGetLastError();
        if (status != hipSuccess)
        {
            ostringstream oss;
            oss << "Kernel launch failed with error: " << hipGetErrorString(status) << "\n";
            throw runtime_error(oss.str());
        }
    }
    cout << "Kernel time: " << timer.tocMs() / kNumTrials << " ms" << endl;
}

int main()
{
    Data data = genData();

    matMulKernel(data);
    matMulCpu(data);

    checkData(data);

    data.free();

    return 0;
}
