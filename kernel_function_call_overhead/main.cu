#include <iostream>

#include "util.cuh"
#include "methods.cuh"

using namespace std;

int kDataSize = 1 << 10;
int kNumCountInc = 1 << 5;
int kNumTrials = 100;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

int main()
{
    long *d_count;
    CHECK_CUDA(hipMalloc(&d_count, kDataSize * sizeof(long)));

    runSetupBaseline(d_count, kDataSize, kNumCountInc, kNumTrials);

    return 0;
}
