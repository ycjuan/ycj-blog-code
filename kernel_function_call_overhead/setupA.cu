#include "hip/hip_runtime.h"
#include <iostream>

#include "util.cuh"
#include "methods.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

__device__ void func0(Param param, size_t taskId)
{
    param.d_count[taskId]++;
}

__device__ void func1(Param param, size_t taskId)
{
    if (taskId != 1)
        func0(param, taskId);
}

__device__ void func2(Param param, size_t taskId)
{
    if (taskId != 2)
        func1(param, taskId);
}

__device__ void func3(Param param, size_t taskId)
{
    if (taskId != 3)
        func2(param, taskId);
}

__device__ void func4(Param param, size_t taskId)
{
    if (taskId != 4)
        func3(param, taskId);
}

__device__ void func5(Param param, size_t taskId)
{
    if (taskId != 5)
        func4(param, taskId);
}

__device__ void func6(Param param, size_t taskId)
{
    if (taskId != 6)
        func5(param, taskId);
}

__device__ void func7(Param param, size_t taskId)
{
    if (taskId != 7)
        func6(param, taskId);
}

__global__ void setupAKernel(Param param)
{
    size_t taskId = (size_t)blockIdx.x * blockDim.x + threadIdx.x;

    if (taskId < param.dataSize)
    {
        param.d_count[taskId] = taskId;
        for (int i = 0; i < param.numCountInc; i++)
        {
            func7(param, taskId);
        }
    }
}

void runSetupA(Param param)
{
    int blockSize = 256;
    int numBlocks = (param.dataSize + blockSize - 1) / blockSize;

    CudaTimer timer;
    timer.tic();
    for (int i = -3; i < param.numTrials; i++)
    {
        setupAKernel<<<numBlocks, blockSize>>>(param);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());
    }
    cout << "setupAKernel time: " << timer.tocMs() << " ms" << endl;
}