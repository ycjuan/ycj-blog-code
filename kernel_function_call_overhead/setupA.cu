#include "hip/hip_runtime.h"
#include <iostream>

#include "util.cuh"
#include "methods.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

__device__ void func0(Param param, size_t wid)
{
    param.d_count[wid] = wid;
    for (int i = 0; i < param.numCountInc; i++)
    {
        param.d_count[wid] += 1;
    }
}

__device__ void func1(Param param, size_t wid)
{
    func0(param, wid);
}

__device__ void func2(Param param, size_t wid)
{
    func1(param, wid);
}

__device__ void func3(Param param, size_t wid)
{
    func2(param, wid);
}

__device__ void func4(Param param, size_t wid)
{
    func3(param, wid);
}

__device__ void func5(Param param, size_t wid)
{
    func4(param, wid);
}

__device__ void func6(Param param, size_t wid)
{
    func5(param, wid);
}

__device__ void func7(Param param, size_t wid)
{
    func6(param, wid);
}

__global__ void setupAKernel(Param param)
{
    size_t wid = (size_t)blockIdx.x * blockDim.x + threadIdx.x;

    if (wid < param.dataSize)
    {
        func7(param, wid);
    }
}

void runSetupA(Param param)
{
    int blockSize = 256;
    int numBlocks = (param.dataSize + blockSize - 1) / blockSize;

    CudaTimer timer;
    timer.tic();
    for (int i = -3; i < param.numTrials; i++)
    {
        setupAKernel<<<numBlocks, blockSize>>>(param);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());
    }
    cout << "setupAKernel time: " << timer.tocMs() << " ms" << endl;
}