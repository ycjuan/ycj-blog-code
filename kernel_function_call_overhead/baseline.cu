#include "hip/hip_runtime.h"
#include <iostream>

#include "util.cuh"
#include "methods.cuh"

using namespace std;

#define CHECK_CUDA(func)                                                                                                           \
    {                                                                                                                              \
        hipError_t status = (func);                                                                                               \
        if (status != hipSuccess)                                                                                                 \
        {                                                                                                                          \
            string error = "CUDA API failed at line " + to_string(__LINE__) + " with error: " + hipGetErrorString(status) + "\n"; \
            throw runtime_error(error);                                                                                            \
        }                                                                                                                          \
    }

__global__ void baselineKernel(Param param)
{
    size_t taskId = (size_t)blockIdx.x * blockDim.x + threadIdx.x;

    if (taskId < param.dataSize)
    {
        param.d_count[taskId] = taskId;
        for (int i = 0; i < param.numCountInc; i++)
        {
            param.d_count[taskId] += 1;
        }
    }
}

void runSetupBaseline(Param param)
{
    int blockSize = 256;
    int numBlocks = (param.dataSize + blockSize - 1) / blockSize;

    CudaTimer timer;
    timer.tic();
    for (int i = -3; i < param.numTrials; i++)
    {
        baselineKernel<<<numBlocks, blockSize>>>(param);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());
    }
    cout << "baselineKernel time: " << timer.tocMs() << " ms" << endl;
}